#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>


/*
    Ensures safe cuda application executions
*/
#define gpuSafeExec(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*
    Clears shared memory which is not full of previous
    numbers. Shmem is remembers values between consecutive
    kernel calls.
*/
__device__ void flushShmem(float *shmem, int shmemSize){
    for (int i = 0; i < shmemSize; i ++)
        shmem[i] = 0.0f;
    return;
}

struct Cluster{
	float* x;
	float* y;
	float* z;
};

void populateClusters(Cluster A, Cluster B, int n) {
	for (int i = 0; i < n; i++) {
		A.x[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
		A.y[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
		A.z[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
		if ((float)rand() / (float)RAND_MAX < 0.01f) {
			B.x[i] = A.x[i] + 10.0f * (float)rand() / (float)RAND_MAX;
			B.y[i] = A.y[i] + 10.0f * (float)rand() / (float)RAND_MAX;
			B.z[i] = A.z[i] + 10.0f * (float)rand() / (float)RAND_MAX;
		}
		else {
            B.x[i] = A.x[i] + 1.0f * (float)rand() / (float)RAND_MAX;
            B.y[i] = A.y[i] + 1.0f * (float)rand() / (float)RAND_MAX;
            B.z[i] = A.z[i] + 1.0f * (float)rand() / (float)RAND_MAX;
		}
	}
}


struct MatrixIndex{
    int i = 0;
    int j = 0;
};

std::ostream& operator<<(std::ostream& os, const MatrixIndex& m){
    os << "(" << m.i << "," << m.j << ")";
    return os;
}

struct ConvertLinearIndexToTriangularMatrixIndex{
    int dim;

    __host__ __device__
    ConvertLinearIndexToTriangularMatrixIndex(int dimension) : dim(dimension){}

    __host__ __device__
    MatrixIndex operator()(int linear) const {
        MatrixIndex result;
       //check if those indices work for you

        //compute i and j from linear index https://stackoverflow.com/questions/27086195/linear-index-upper-triangular-matrix
        result.i = dim - 2 - floor(sqrt(-8*linear + 4*dim*(dim-1)-7)/2.0 - 0.5);
        result.j = linear + result.i + 1 - dim*(dim-1)/2 + (dim-result.i)*((dim-result.i)-1)/2;

        return result;
    }
};

struct ComputeDelta{
    Cluster A;
    Cluster B;

    __host__ __device__
    ComputeDelta(Cluster _A, Cluster _B){
        /* init A and B*/
        A = _A;
        B = _B;
    }

    __host__ __device__
    float operator()(const MatrixIndex& index) const{
        
        float da = 0;
        float db = 0;

        da = sqrt((A.x[index.i]-A.x[index.j])*(A.x[index.i]-A.x[index.j])
                    + (A.y[index.i]-A.y[index.j])*(A.y[index.i]-A.y[index.j])
                    + (A.z[index.i]-A.z[index.j])*(A.z[index.i]-A.z[index.j]));
        db = sqrt((B.x[index.i]-B.x[index.j])*(B.x[index.i]-B.x[index.j])
                    + (B.y[index.i]-B.y[index.j])*(B.y[index.i]-B.y[index.j])
                    + (B.z[index.i]-B.z[index.j])*(B.z[index.i]-B.z[index.j]));

        return (da-db) * (da-db);
    }
};

float solveGPU_thrust(Cluster A, Cluster B, int n) {
    const int dim = n;
    const int elems = round(n*(n-1)/2); //upper triangular(without diagonal) number of elements formula
    auto matrixIndexIterator = thrust::make_transform_iterator(
        thrust::make_counting_iterator(0),
        ConvertLinearIndexToTriangularMatrixIndex{dim}
    );


    //for(int i = 0; i < elems; i++){
    //    std::cout << matrixIndexIterator[i] << " ";
    //}
    
    float result = thrust::transform_reduce(
        matrixIndexIterator, 
        matrixIndexIterator + elems, 
        ComputeDelta{A,B}, 
        float(0), 
        thrust::plus<float>{}
    );
    return sqrt(1/((float)n*((float)n-1)) * result);
}

float solveCPU(Cluster A, Cluster B, int n) {
	float difference = 0.0f;
	for (int i = 0; i < n-1; i++) {
		float tmp = 0.0f;
		for (int j = i+1; j < n; j++) {
			float diff_a = sqrt((A.x[i]-A.x[j])*(A.x[i]-A.x[j])
				+ (A.y[i]-A.y[j])*(A.y[i]-A.y[j])
				+ (A.z[i]-A.z[j])*(A.z[i]-A.z[j]));
			float diff_b = sqrt((B.x[i]-B.x[j])*(B.x[i]-B.x[j])
				+ (B.y[i]-B.y[j])*(B.y[i]-B.y[j])
				+ (B.z[i]-B.z[j])*(B.z[i]-B.z[j]));
			tmp += (diff_a-diff_b) * (diff_a-diff_b);
		}
		difference += tmp;
	}
	return sqrt(1/((float)n*((float)n-1)) * difference);
}

int main(int argc, char **argv){
    constexpr int N = 1024*64;

	Cluster A, B;
	A.x = A.y = A.z = B.x = B.y = B.z = NULL;
	Cluster dA, dB;
	dA.x = dA.y = dA.z = dB.x = dB.y = dB.z = NULL;
	float diff_CPU, diff_GPU;

	// parse command line
	int device = 0;
	if (argc == 2) 
		device = atoi(argv[1]);
	if (hipSetDevice(device) != hipSuccess){
		fprintf(stderr, "Cannot set CUDA device!\n");
		exit(1);
	}

	printf("Number of points per cluster: %d\n", N);
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("Using device %d: \"%s\"\n", device, deviceProp.name);
	//printf("%d \n",*deviceProp.maxGridSize);

	// create events for timing
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

	// allocate and set host memory
	A.x = (float*)malloc(N*sizeof(A.x[0]));
	A.y = (float*)malloc(N*sizeof(A.y[0]));
	A.z = (float*)malloc(N*sizeof(A.z[0]));
	B.x = (float*)malloc(N*sizeof(B.x[0]));
    B.y = (float*)malloc(N*sizeof(B.y[0]));
    B.z = (float*)malloc(N*sizeof(B.z[0]));
	populateClusters(A, B, N);      
 
	// allocate and set device memory
	if (hipMalloc((void**)&dA.x, N*sizeof(dA.x[0])) != hipSuccess
	|| hipMalloc((void**)&dA.y, N*sizeof(dA.y[0])) != hipSuccess
	|| hipMalloc((void**)&dA.z, N*sizeof(dA.z[0])) != hipSuccess
	|| hipMalloc((void**)&dB.x, N*sizeof(dB.x[0])) != hipSuccess
    || hipMalloc((void**)&dB.y, N*sizeof(dB.y[0])) != hipSuccess
    || hipMalloc((void**)&dB.z, N*sizeof(dB.z[0])) != hipSuccess) {
		fprintf(stderr, "Device memory allocation error!\n");
		hipError_t err = hipGetLastError();
		if (err != hipSuccess){
			printf("CUDA ERROR while executing the kernel: %s\n",hipGetErrorString(err));
			return 103;
		}
		goto cleanup;
	}
	hipMemcpy(dA.x, A.x, N*sizeof(dA.x[0]), hipMemcpyHostToDevice);
	hipMemcpy(dA.y, A.y, N*sizeof(dA.y[0]), hipMemcpyHostToDevice);
	hipMemcpy(dA.z, A.z, N*sizeof(dA.z[0]), hipMemcpyHostToDevice);
	hipMemcpy(dB.x, B.x, N*sizeof(dB.x[0]), hipMemcpyHostToDevice);
    hipMemcpy(dB.y, B.y, N*sizeof(dB.y[0]), hipMemcpyHostToDevice);
    hipMemcpy(dB.z, B.z, N*sizeof(dB.z[0]), hipMemcpyHostToDevice);

	// solve on CPU
    printf("Solving on CPU...\n");
	hipEventRecord(start, 0);
	diff_CPU = solveCPU(A, B, N);
	hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("CPU performance: %f megapairs/s\n",
        float(N)*float(N-1)/2.0f/time/1e3f);

	
    // solve on GPU with thrust
	printf("Solving on GPU with thrust...\n");
	hipEventRecord(start, 0);
	// run it 10x for more accurately timing results
    for (int i = 0; i < 10; i++){
		diff_GPU = solveGPU_thrust(dA, dB, N);

	}
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
	printf("GPU performance with thrust: %f megapairs/s\n",
        float(N)*float(N-1)/2.0f/time/1e2f);
    printf("time:%f\n",time);

	printf("CPU diff: %f\nGPU diff: %f\n", diff_CPU, diff_GPU);
	// check GPU results
	if ( fabsf((diff_CPU-diff_GPU) / ((diff_CPU+diff_GPU)/2.0f)) < 0.01f)
		printf("Test OK :-).\n");
	else
		 fprintf(stderr, "Data mismatch: %f should be %f :-(\n", diff_GPU, diff_CPU);

cleanup:
	hipEventDestroy(start);
    hipEventDestroy(stop);
    
	if (dA.x) hipFree(dA.x);
	if (dA.y) hipFree(dA.y);
	if (dA.z) hipFree(dA.z);
	if (dB.x) hipFree(dB.x);
    if (dB.y) hipFree(dB.y);
    if (dB.z) hipFree(dB.z);
	if (A.x) free(A.x);
	if (A.y) free(A.y);
	if (A.z) free(A.z);
	if (B.x) free(B.x);
    if (B.y) free(B.y);
    if (B.z) free(B.z);

	return 0;
}