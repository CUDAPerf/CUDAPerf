#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>

#include <hipcub/hipcub.hpp>

/*
    Ensures safe cuda application executions
*/
#define gpuSafeExec(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*
    Clears shared memory which is not full of previous
    numbers. Shmem is remembers values between consecutive
    kernel calls.
*/
__device__ void flushShmem(float *shmem, int shmemSize){
    for (int i = 0; i < shmemSize; i ++)
        shmem[i] = 0.0f;
    return;
}

struct Cluster{
	float* x;
	float* y;
	float* z;
};

void populateClusters(Cluster A, Cluster B, int n) {
	for (int i = 0; i < n; i++) {
		A.x[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
		A.y[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
		A.z[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
		if ((float)rand() / (float)RAND_MAX < 0.01f) {
			B.x[i] = A.x[i] + 10.0f * (float)rand() / (float)RAND_MAX;
			B.y[i] = A.y[i] + 10.0f * (float)rand() / (float)RAND_MAX;
			B.z[i] = A.z[i] + 10.0f * (float)rand() / (float)RAND_MAX;
		}
		else {
            B.x[i] = A.x[i] + 1.0f * (float)rand() / (float)RAND_MAX;
            B.y[i] = A.y[i] + 1.0f * (float)rand() / (float)RAND_MAX;
            B.z[i] = A.z[i] + 1.0f * (float)rand() / (float)RAND_MAX;
		}
	}
}


struct MatrixIndex{
    int i = 0;
    int j = 0;
};

std::ostream& operator<<(std::ostream& os, const MatrixIndex& m){
    os << "(" << m.i << "," << m.j << ")";
    return os;
}

struct ConvertLinearIndexToTriangularMatrixIndex{
    int dim;

    __host__ __device__
    ConvertLinearIndexToTriangularMatrixIndex(int dimension) : dim(dimension){}

    __host__ __device__
    MatrixIndex operator()(int linear) const {
        MatrixIndex result;
       //check if those indices work for you

        //compute i and j from linear index https://stackoverflow.com/questions/27086195/linear-index-upper-triangular-matrix
        result.i = dim - 2 - floor(sqrt(-8*linear + 4*dim*(dim-1)-7)/2.0 - 0.5);
        result.j = linear + result.i + 1 - dim*(dim-1)/2 + (dim-result.i)*((dim-result.i)-1)/2;

        return result;
    }
};

struct ComputeDelta{
    Cluster A;
    Cluster B;

    __host__ __device__
    ComputeDelta(Cluster _A, Cluster _B){
        /* init A and B*/
        A = _A;
        B = _B;
    }

    __host__ __device__
    float operator()(const MatrixIndex& index) const{
        
        float da = 0;
        float db = 0;

        da = sqrt((A.x[index.i]-A.x[index.j])*(A.x[index.i]-A.x[index.j])
                    + (A.y[index.i]-A.y[index.j])*(A.y[index.i]-A.y[index.j])
                    + (A.z[index.i]-A.z[index.j])*(A.z[index.i]-A.z[index.j]));
        db = sqrt((B.x[index.i]-B.x[index.j])*(B.x[index.i]-B.x[index.j])
                    + (B.y[index.i]-B.y[index.j])*(B.y[index.i]-B.y[index.j])
                    + (B.z[index.i]-B.z[index.j])*(B.z[index.i]-B.z[index.j]));

        return (da-db) * (da-db);
    }
};

float solveGPU_thrust(Cluster A, Cluster B, int n) {
    const int dim = n;
    const int elems = round(n*(n-1)/2); //upper triangular(without diagonal) number of elements formula
    auto matrixIndexIterator = thrust::make_transform_iterator(
        thrust::make_counting_iterator(0),
        ConvertLinearIndexToTriangularMatrixIndex{dim}
    );


    //for(int i = 0; i < elems; i++){
    //    std::cout << matrixIndexIterator[i] << " ";
    //}
    
    float result = thrust::transform_reduce(
        matrixIndexIterator, 
        matrixIndexIterator + elems, 
        ComputeDelta{A,B}, 
        float(0), 
        thrust::plus<float>{}
    );
    return sqrt(1/((float)n*((float)n-1)) * result);
}




__global__ void cluster_similarity_reduction(const Cluster A, const Cluster B, const int n , float* output, int shmemSize) {
    extern __shared__ float sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    //clear SHMEM
    if (tid == 0)
    {
        flushShmem(sdata, shmemSize);
    }

    //wait for shem flush
    __syncthreads();
    
    //do the math
    for(int j = i+1; j < n; j++){
        float da = sqrt((A.x[i]-A.x[j])*(A.x[i]-A.x[j])
                    + (A.y[i]-A.y[j])*(A.y[i]-A.y[j])
                    + (A.z[i]-A.z[j])*(A.z[i]-A.z[j]));
        float db = sqrt((B.x[i]-B.x[j])*(B.x[i]-B.x[j])
                    + (B.y[i]-B.y[j])*(B.y[i]-B.y[j])
                    + (B.z[i]-B.z[j])*(B.z[i]-B.z[j]));
       //float da = norm3df(A.x[i]-A.x[j], A.y[i]-A.y[j], A.z[i]-A.z[j]);
      // float db = norm3df(B.x[i]-B.x[j], B.y[i]-B.y[j], B.z[i]-B.z[j]);
        sdata[tid] += (da-db) * (da-db);
    }

    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            sdata[tid] += sdata[tid + s];
        }

        __syncthreads();
    }
    //write result of this block to global memory
    if (tid == 0) output[blockIdx.x] = sdata[0];
}


float solveGPU(Cluster clusterA, Cluster clusterB, int n) {
    float *hostOutput; 
    float *deviceOutput; 

    int blockSize;
    int minGridSize;
    int gridSize;

    //use cuda occupancy calculator to determine grid and block sizes
    gpuSafeExec(hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, 
                                       cluster_similarity_reduction, 0, 0)); 

    //determine correct number of output elements after reduction
    int numOutputElements = n / (blockSize / 2);
    if (n % (blockSize / 2)) {
        numOutputElements++;
    }

    hostOutput = (float *)malloc(numOutputElements * sizeof(float));
    // Round up according to array size 
    gridSize = (n + blockSize - 1) / blockSize; 

    //allocate GPU memory
    gpuSafeExec(hipMalloc((void **)&deviceOutput, numOutputElements * sizeof(float)));
    //std::cerr << "cluster_similarity_reduction<<<" << gridSize << "," << blockSize << "," << blockSize*sizeof(float) << ">>>\n";
    cluster_similarity_reduction <<<gridSize, blockSize, blockSize*sizeof(float) >>>(clusterA, clusterB, n, deviceOutput, blockSize);
    //move GPU results to CPU via PCIe
    gpuSafeExec(hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost));

    //accumulate the sum in the first element
    for (int i = 1; i < numOutputElements; i++) {
        hostOutput[0] += hostOutput[i]; 
    }
    
    //use overall square root out of GPU, to avoid race condition
    float retval = sqrt(1/((float)n*((float)n-1)) * hostOutput[0]);

    //cleanup
    gpuSafeExec(hipFree(deviceOutput));
    free(hostOutput);

    return retval;
}


template<int blocksize>
__global__ void cluster_similarity_reduction_cubreduce(const Cluster A, const Cluster B, const int n , float* output, int shmemSize) {

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    //do the math
    float mysum = 0.0f;
    for(int j = i+1; j < n; j++){
        float da = sqrt((A.x[i]-A.x[j])*(A.x[i]-A.x[j])
                    + (A.y[i]-A.y[j])*(A.y[i]-A.y[j])
                    + (A.z[i]-A.z[j])*(A.z[i]-A.z[j]));
        float db = sqrt((B.x[i]-B.x[j])*(B.x[i]-B.x[j])
                    + (B.y[i]-B.y[j])*(B.y[i]-B.y[j])
                    + (B.z[i]-B.z[j])*(B.z[i]-B.z[j]));
        mysum += (da-db) * (da-db);
    }

    using BlockReduce = hipcub::BlockReduce<float, blocksize>;
    __shared__ typename BlockReduce::TempStorage tmp;

    float sum = BlockReduce(tmp).Sum(mysum);

    //write result of this block to global memory
    if (threadIdx.x == 0){
        output[blockIdx.x] = sum;
    }
}



float solveGPU_cubblockreduce(Cluster clusterA, Cluster clusterB, int n) {
    float *hostOutput; 
    float *deviceOutput; 

    int gridSize;

    constexpr int blockSize = 256;
    auto kernel = cluster_similarity_reduction_cubreduce<blockSize>;

    int deviceId = 0;
    int numSMs = 0;
    int maxBlocksPerSM = 0;
    gpuSafeExec(hipGetDevice(&deviceId));
    gpuSafeExec(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, deviceId));
    gpuSafeExec(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &maxBlocksPerSM,
        kernel,
        blockSize, 
        0
    ));

    //determine correct number of output elements after reduction
    int numOutputElements = n / (blockSize / 2);
    if (n % (blockSize / 2)) {
        numOutputElements++;
    }

    hostOutput = (float *)malloc(numOutputElements * sizeof(float));
    // Round up according to array size 
    gridSize = (n + blockSize - 1) / blockSize; 

    //allocate GPU memory
    gpuSafeExec(hipMalloc((void **)&deviceOutput, numOutputElements * sizeof(float)));
    //std::cerr << "cluster_similarity_reduction_cubreduce<<<" << gridSize << "," << blockSize << "," << 0 << ">>>\n";
    kernel<<<gridSize, blockSize, 0 >>>(clusterA, clusterB, n, deviceOutput, blockSize);
    //move GPU results to CPU via PCIe
    gpuSafeExec(hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost));

    //accumulate the sum in the first element
    for (int i = 1; i < numOutputElements; i++) {
        hostOutput[0] += hostOutput[i]; 
    }
    
    //use overall square root out of GPU, to avoid race condition
    float retval = sqrt(1/((float)n*((float)n-1)) * hostOutput[0]);

    //cleanup
    gpuSafeExec(hipFree(deviceOutput));
    free(hostOutput);

    return retval;
}





float solveCPU(Cluster A, Cluster B, int n) {
	float difference = 0.0f;
	for (int i = 0; i < n-1; i++) {
		float tmp = 0.0f;
		for (int j = i+1; j < n; j++) {
			float diff_a = sqrt((A.x[i]-A.x[j])*(A.x[i]-A.x[j])
				+ (A.y[i]-A.y[j])*(A.y[i]-A.y[j])
				+ (A.z[i]-A.z[j])*(A.z[i]-A.z[j]));
			float diff_b = sqrt((B.x[i]-B.x[j])*(B.x[i]-B.x[j])
				+ (B.y[i]-B.y[j])*(B.y[i]-B.y[j])
				+ (B.z[i]-B.z[j])*(B.z[i]-B.z[j]));
			tmp += (diff_a-diff_b) * (diff_a-diff_b);
		}
		difference += tmp;
	}
	return sqrt(1/((float)n*((float)n-1)) * difference);
}

int main(int argc, char **argv){
    constexpr int N = 1024*64;

	Cluster A, B;
	A.x = A.y = A.z = B.x = B.y = B.z = NULL;
	Cluster dA, dB;
	dA.x = dA.y = dA.z = dB.x = dB.y = dB.z = NULL;
	float diff_CPU, diff_GPU;

	// parse command line
	int device = 0;
	if (argc == 2) 
		device = atoi(argv[1]);
	if (hipSetDevice(device) != hipSuccess){
		fprintf(stderr, "Cannot set CUDA device!\n");
		exit(1);
	}

	printf("Number of points per cluster: %d\n", N);
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    printf("Using device %d: \"%s\"\n", device, deviceProp.name);
	//printf("%d \n",*deviceProp.maxGridSize);

	// create events for timing
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

	// allocate and set host memory
	A.x = (float*)malloc(N*sizeof(A.x[0]));
	A.y = (float*)malloc(N*sizeof(A.y[0]));
	A.z = (float*)malloc(N*sizeof(A.z[0]));
	B.x = (float*)malloc(N*sizeof(B.x[0]));
    B.y = (float*)malloc(N*sizeof(B.y[0]));
    B.z = (float*)malloc(N*sizeof(B.z[0]));
	populateClusters(A, B, N);      
 
	// allocate and set device memory
	if (hipMalloc((void**)&dA.x, N*sizeof(dA.x[0])) != hipSuccess
	|| hipMalloc((void**)&dA.y, N*sizeof(dA.y[0])) != hipSuccess
	|| hipMalloc((void**)&dA.z, N*sizeof(dA.z[0])) != hipSuccess
	|| hipMalloc((void**)&dB.x, N*sizeof(dB.x[0])) != hipSuccess
    || hipMalloc((void**)&dB.y, N*sizeof(dB.y[0])) != hipSuccess
    || hipMalloc((void**)&dB.z, N*sizeof(dB.z[0])) != hipSuccess) {
		fprintf(stderr, "Device memory allocation error!\n");
		hipError_t err = hipGetLastError();
		if (err != hipSuccess){
			printf("CUDA ERROR while executing the kernel: %s\n",hipGetErrorString(err));
			return 103;
		}
		goto cleanup;
	}
	hipMemcpy(dA.x, A.x, N*sizeof(dA.x[0]), hipMemcpyHostToDevice);
	hipMemcpy(dA.y, A.y, N*sizeof(dA.y[0]), hipMemcpyHostToDevice);
	hipMemcpy(dA.z, A.z, N*sizeof(dA.z[0]), hipMemcpyHostToDevice);
	hipMemcpy(dB.x, B.x, N*sizeof(dB.x[0]), hipMemcpyHostToDevice);
    hipMemcpy(dB.y, B.y, N*sizeof(dB.y[0]), hipMemcpyHostToDevice);
    hipMemcpy(dB.z, B.z, N*sizeof(dB.z[0]), hipMemcpyHostToDevice);

	// solve on CPU
    printf("Solving on CPU...\n");
	hipEventRecord(start, 0);
	diff_CPU = solveCPU(A, B, N);
	hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    printf("CPU performance: %f megapairs/s\n",
        float(N)*float(N-1)/2.0f/time/1e3f);

	// solve on GPU
    
	printf("Solving on GPU with default kernel...\n");
	hipEventRecord(start, 0);
	// run it 10x for more accurately timing results
    for (int i = 0; i < 10; i++){
		diff_GPU = solveGPU(dA, dB, N);

	}
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
	printf("GPU performance: %f megapairs/s\n",
        float(N)*float(N-1)/2.0f/time/1e2f);
    printf("time:%f\n",time);

	printf("CPU diff: %f\nGPU diff: %f\n", diff_CPU, diff_GPU);
	// check GPU results
	if ( fabsf((diff_CPU-diff_GPU) / ((diff_CPU+diff_GPU)/2.0f)) < 0.01f)
		printf("Test OK :-).\n");
	else
		 fprintf(stderr, "Data mismatch: %f should be %f :-(\n", diff_GPU, diff_CPU);

    

    // solve on GPU with cub block reduce
	printf("Solving on GPU with cub block reduce kernel...\n");
	hipEventRecord(start, 0);
	// run it 10x for more accurately timing results
    for (int i = 0; i < 10; i++){
		diff_GPU = solveGPU_cubblockreduce(dA, dB, N);

	}
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
	printf("GPU performance with cub blockreduce: %f megapairs/s\n",
        float(N)*float(N-1)/2.0f/time/1e2f);
    printf("time:%f\n",time);

	printf("CPU diff: %f\nGPU diff: %f\n", diff_CPU, diff_GPU);
	// check GPU results
	if ( fabsf((diff_CPU-diff_GPU) / ((diff_CPU+diff_GPU)/2.0f)) < 0.01f)
		printf("Test OK :-).\n");
	else
		 fprintf(stderr, "Data mismatch: %f should be %f :-(\n", diff_GPU, diff_CPU);




    // solve on GPU
	printf("Solving on GPU with thrust...\n");
	hipEventRecord(start, 0);
	// run it 10x for more accurately timing results
    for (int i = 0; i < 10; i++){
		diff_GPU = solveGPU_thrust(dA, dB, N);

	}
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
	printf("GPU performance with thrust: %f megapairs/s\n",
        float(N)*float(N-1)/2.0f/time/1e2f);
    printf("time:%f\n",time);

	printf("CPU diff: %f\nGPU diff: %f\n", diff_CPU, diff_GPU);
	// check GPU results
	if ( fabsf((diff_CPU-diff_GPU) / ((diff_CPU+diff_GPU)/2.0f)) < 0.01f)
		printf("Test OK :-).\n");
	else
		 fprintf(stderr, "Data mismatch: %f should be %f :-(\n", diff_GPU, diff_CPU);

cleanup:
	hipEventDestroy(start);
    hipEventDestroy(stop);
    
	if (dA.x) hipFree(dA.x);
	if (dA.y) hipFree(dA.y);
	if (dA.z) hipFree(dA.z);
	if (dB.x) hipFree(dB.x);
    if (dB.y) hipFree(dB.y);
    if (dB.z) hipFree(dB.z);
	if (A.x) free(A.x);
	if (A.y) free(A.y);
	if (A.z) free(A.z);
	if (B.x) free(B.x);
    if (B.y) free(B.y);
    if (B.z) free(B.z);

	return 0;
}