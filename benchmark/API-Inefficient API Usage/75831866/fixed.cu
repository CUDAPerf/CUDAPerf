#include <hip/hip_runtime.h>
#include <string>
#include <chrono>
#include <random>
#include <iostream>
using namespace std;

  int N = 1024*1024;

class MyTimer {
    std::chrono::time_point<std::chrono::system_clock> start;

public:
    void startCounter() {
        start = std::chrono::system_clock::now();
    }

    int64_t getCounterNs() {
        return std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now() - start).count();
    }

    int64_t getCounterMs() {
        return std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now() - start).count();
    }

    double getCounterMsPrecise() {
        return std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now() - start).count()
                / 1000000.0;
    }
};

__global__
void HelloWorld()
{
  printf("Hello world\n");
}

volatile double dummy = 0;

__global__
void multiply1(int N, float* __restrict__ output, const float* __restrict__ x, const float* __restrict__ y)
{
  int start = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = start; i < N; i += stride) {
    output[i] = x[i] * y[i];
  }
}


int test1()
{
  MyTimer timer;
  srand(time(NULL));
  HelloWorld<<<1,1>>>();
  timer.startCounter();
  float *d_a, *d_b, *d_c;
#ifdef USE_MANAGED
  cudaMallocManaged(&d_a, N * sizeof(float));
  cudaMallocManaged(&d_b, N * sizeof(float));
  cudaMallocManaged(&d_c, N * sizeof(float));
  for (int i = 0; i < N; i++) {
    d_a[i] = float(rand() % 1000000) / (rand() % 1000 + 1);
    d_b[i] = float(rand() % 1000000) / (rand() % 1000 + 1);
    d_c[i] = 0.f;
  }
  cudaMemPrefetchAsync(d_a, N*sizeof(float), 0);
  cudaMemPrefetchAsync(d_b, N*sizeof(float), 0);
  cudaMemPrefetchAsync(d_c, N*sizeof(float), 0);
#else
  float* h_a = new float[N];
  float* h_b = new float[N];
  float* h_res = new float[N];
  for (int i = 0; i < N; i++) {
    h_a[i] = float(rand() % 1000000) / (rand() % 1000 + 1);
    h_b[i] = float(rand() % 1000000) / (rand() % 1000 + 1);
  }
  hipMalloc(&d_a, N * sizeof(float));
  hipMalloc(&d_b, N * sizeof(float));
  hipMalloc(&d_c, N * sizeof(float));
  hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
#endif
  hipDeviceSynchronize();
  dummy = timer.getCounterMsPrecise();
  cout << "alloc/H2D cost = " << dummy << "\n";
  constexpr int GRID_DIM = 256;
  constexpr int BLOCK_DIM = 256;

  timer.startCounter();
  multiply1<<<GRID_DIM, BLOCK_DIM>>>(N, d_c, d_a, d_b);
  hipDeviceSynchronize();
  dummy = timer.getCounterMsPrecise();
  cout << "kernel cost = " << dummy << "\n";
  float *res = d_c;
  float *a = d_a;
  float *b = d_b;
#ifndef USE_MANAGED
  timer.startCounter();
  hipMemcpy(h_res, d_c, N * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  dummy = timer.getCounterMsPrecise();
  cout << "D2H copy cost = " << timer.getCounterMsPrecise() << "\n";
  res = h_res;
  a = h_a;
  b = h_b;
#endif

  for (int i = 0; i < N; i++) if (res[i] != (a[i]*b[i])) {
    cout << "error\n";
    exit(1);
  }
  return 0;
}


int main(){
  for(int i = 0; i < 10; i++){
    test1();
  }
}