#include "hipblas.h"
#include "hip/hip_runtime.h"
#include <iostream>

#define TILE_WIDTH 32
const int Width = 1024;

void testCuBLASMatrixMulKernel(const int Width) {
  float *A_h = (float *)malloc(Width * Width * sizeof(float));
  float *B_h = (float *)malloc(Width * Width * sizeof(float));
  for (auto i = 0; i < Width * Width; i++) {
    A_h[i] = 1.0 * float(i) / 4.0;
    B_h[i] = 2.0 * float(i) / 4.0;
  }
  float *A_d, *B_d, *C_d;
  hipMalloc((void **)&A_d, Width * Width * sizeof(float));
  hipMalloc((void **)&B_d, Width * Width * sizeof(float));
  hipMalloc((void **)&C_d, Width * Width * sizeof(float));

  hipMemcpy(A_d, A_h, Width * Width * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, Width * Width * sizeof(float), hipMemcpyHostToDevice);

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  const float alpha = 1.0f;
  const float beta = 0.0f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, Width, Width, Width, &alpha, A_d, Width, B_d, Width, &beta, C_d, Width);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0.0;
  hipEventElapsedTime(&milliseconds, start, stop);

  std::cout << "Elapsed time for testCuBLASMatrixMulKernel : " << milliseconds
            << '\n';

  hipblasDestroy(handle);
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  free(A_h);
  free(B_h);
}

int main(int argc, const char* argv[]) {
	hipDeviceSynchronize();
	std::cout << "Arithmetic Intensity : " << 1.0 / 3.0 * float(Width) << '\n';
  for(int i=0;i<1;i++){
    testCuBLASMatrixMulKernel(Width);
  }
	return 0;
}