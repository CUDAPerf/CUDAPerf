#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

#define TILE_SIZE 8

// Check CUDA errors
#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Tiled Matrix Multiplication Kernel
__global__
void tiledMatMult_kernel(float* Cd, float* Ad, float* Bd, int width) {
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float res = 0;

    for (int i = 0; i < ceil((float)width / TILE_SIZE); ++i) {
        if (tx + i * TILE_SIZE < width && row < width) {
            tileA[ty][tx] = Ad[row * width + tx + i * TILE_SIZE];
        } else {
            tileA[ty][tx] = 0;
        }

        if (ty + i * TILE_SIZE < width && col < width) {
            tileB[ty][tx] = Bd[(ty + i * TILE_SIZE) * width + col];
        } else {
            tileB[ty][tx] = 0;
        }
        __syncthreads();

        for (int k = 0; k < TILE_SIZE; k++) {
            res += tileA[ty][k] * tileB[k][tx];
        }
        __syncthreads();
    }

    if (col < width && row < width) {
        Cd[row * width + col] = res;
    }
}

// Simple Matrix Multiplication Kernel
__global__
void matMultKer(float* Cd, float* Ad, float* Bd, int width) {
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int col = threadIdx.x + blockDim.x * blockIdx.x;

    if (row < width && col < width) {
        float res = 0;
        for (int k = 0; k < width; ++k) {
            res += Ad[row * width + k] * Bd[k * width + col];
        }
        Cd[row * width + col] = res;
    }
}

// Host code to initialize matrices and compare kernel execution time
void initializeMatrix(float* matrix, int width) {
    for (int i = 0; i < width * width; ++i) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

void compareKernels(int width) {
    // Host matrices
    float *A, *B, *C_tiled, *C_basic;
    int size = width * width * sizeof(float);

    // Allocate host memory
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C_tiled = (float*)malloc(size);
    C_basic = (float*)malloc(size);

    // Initialize matrices
    initializeMatrix(A, width);
    initializeMatrix(B, width);

    // Device matrices
    float *Ad, *Bd, *Cd_tiled, *Cd_basic;

    // Allocate device memory
    CUDA_CHECK(hipMalloc(&Ad, size));
    CUDA_CHECK(hipMalloc(&Bd, size));
    CUDA_CHECK(hipMalloc(&Cd_tiled, size));
    CUDA_CHECK(hipMalloc(&Cd_basic, size));

    // Copy matrices to device
    CUDA_CHECK(hipMemcpy(Ad, A, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(Bd, B, size, hipMemcpyHostToDevice));

    // Define block and grid dimensions
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    dim3 dimGrid((width + TILE_SIZE - 1) / TILE_SIZE, (width + TILE_SIZE - 1) / TILE_SIZE);

    // Time the tiled kernel
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    
    CUDA_CHECK(hipEventRecord(start));
    for(int i=0;i<10;i++){
        tiledMatMult_kernel<<<dimGrid, dimBlock>>>(Cd_tiled, Ad, Bd, width);
    }
    CUDA_CHECK(hipEventRecord(stop));

    CUDA_CHECK(hipEventSynchronize(stop));
    float tiledTime;
    CUDA_CHECK(hipEventElapsedTime(&tiledTime, start, stop));

    // Time the basic kernel
    CUDA_CHECK(hipEventRecord(start));
    for(int i=0;i<10;i++){
        matMultKer<<<dimGrid, dimBlock>>>(Cd_basic, Ad, Bd, width);
    }
    CUDA_CHECK(hipEventRecord(stop));

    CUDA_CHECK(hipEventSynchronize(stop));
    float basicTime;
    CUDA_CHECK(hipEventElapsedTime(&basicTime, start, stop));

    // Copy results back to host
    CUDA_CHECK(hipMemcpy(C_tiled, Cd_tiled, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(C_basic, Cd_basic, size, hipMemcpyDeviceToHost));

    // Compare execution times
    std::cout << "Tiled Kernel Time: " << tiledTime << " ms\n";
    std::cout << "Basic Kernel Time: " << basicTime << " ms\n";

    // Free memory
    free(A);
    free(B);
    free(C_tiled);
    free(C_basic);
    CUDA_CHECK(hipFree(Ad));
    CUDA_CHECK(hipFree(Bd));
    CUDA_CHECK(hipFree(Cd_tiled));
    CUDA_CHECK(hipFree(Cd_basic));
}

int main() {
    srand(static_cast<unsigned int>(time(0)));

    int width = 1024; // Set matrix size (width x width)
    compareKernels(width);

    return 0;
}