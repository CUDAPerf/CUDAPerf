#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/iterator/zip_iterator.h>

const int N = 1024;

__global__ void forward_pass(double* w, double* b){
    int idx = threadIdx.x;
    __shared__ double w_buffer[N];
    __shared__ double b_buffer[N];
    w_buffer[idx] = w[idx];
    b_buffer[idx] = b[idx];
    __syncthreads();
    if (idx == 0) {
        for (int i = 1; i < N; i++) {
            b_buffer[i] = b_buffer[i] - b_buffer[i - 1] * w_buffer[i];
        }
    }
    __syncthreads();
    b[idx] = b_buffer[idx];
}

struct scan_op {
    template <typename T1, typename T2>
    __host__ __device__
    T1 operator()(const T1 &t1, const T2 &t2) {
        T1 ret;
        thrust::get<0>(ret) = thrust::get<0>(t1) * thrust::get<0>(t2);
        thrust::get<1>(ret) = thrust::get<1>(t1) * thrust::get<0>(t2) + thrust::get<1>(t2);
        return ret;
    }
};

using mt = double;
using namespace thrust::placeholders;

int main() {
    //hipSetDevice(1);
    mt *h_w, *d_w, *h_b, *d_b, *h_r;
    h_w = new mt[N];
    h_r = new mt[N];
    h_b = new mt[N];

    hipMalloc(&d_b, N * sizeof(d_b[0]));
    hipMalloc(&d_w, N * sizeof(d_w[0]));
    for (int i = 0; i < N; i++) {
        h_w[i] = rand() / (double)RAND_MAX;
        h_b[i] = rand() / (double)RAND_MAX;
    }
    hipMemcpy(d_b, h_b, N * sizeof(d_b[0]), hipMemcpyHostToDevice);
    hipMemcpy(d_w, h_w, N * sizeof(d_w[0]), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    forward_pass<<<1, N>>>(d_w, d_b);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

    hipMemcpy(h_r, d_b, N * sizeof(d_b[0]), hipMemcpyDeviceToHost);
    for (int i = 0; i < 8; i++) std::cout << h_r[i] << ",";
    std::cout << std::endl;

    // Thrust
    thrust::device_vector<mt> db(h_b, h_b + N);
    thrust::device_vector<mt> da(h_w, h_w + N);
    thrust::transform(da.begin(), da.end(), da.begin(), _1 * (-1));
    thrust::device_vector<mt> dy(N);
    thrust::device_vector<mt> dx(N);

    hipEventRecord(start);
    thrust::inclusive_scan(
        thrust::make_zip_iterator(thrust::make_tuple(da.begin(), db.begin())),
        thrust::make_zip_iterator(thrust::make_tuple(da.end(), db.end())),
        thrust::make_zip_iterator(thrust::make_tuple(dy.begin(), dx.begin())),
        scan_op()
    );
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Thrust execution time: " << milliseconds << " ms" << std::endl;

    thrust::host_vector<mt> hx = dx;
    thrust::copy_n(hx.begin(), 8, std::ostream_iterator<mt>(std::cout, ","));
    std::cout << std::endl;

    // free
    hipFree(d_b);
    hipFree(d_w);
    delete[] h_w;
    delete[] h_b;
    delete[] h_r;

    return 0;
}