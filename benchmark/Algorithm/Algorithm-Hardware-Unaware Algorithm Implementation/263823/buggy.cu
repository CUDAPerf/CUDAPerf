#include "hipblas.h"
#include "hip/hip_runtime.h"
#include <iostream>

#define TILE_WIDTH 32
const int Width = 1024;

__global__ void MatrixMulKernel(const float *M, const float *N, float *P,
                                const int width) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int column = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < width && column < width) {
    float Pvalue = .0;
    for (int k = 0; k < width; ++k) {
      Pvalue += M[row * width + k] * N[k * width + column];
    }
    P[row * width + column] = Pvalue;
  }
}

void testMatrixMulKernel(const int Width) {
  float *A_h = (float *)malloc(Width * Width * sizeof(float));
  float *B_h = (float *)malloc(Width * Width * sizeof(float));
  for (auto i = 0; i < Width * Width; i++) {
    A_h[i] = 1.0 * float(i) / 4.0;
    B_h[i] = 2.0 * float(i) / 4.0;
  }
  float *A_d, *B_d, *C_d;
  hipMalloc((void **)&A_d, Width * Width * sizeof(float));
  hipMalloc((void **)&B_d, Width * Width * sizeof(float));
  hipMalloc((void **)&C_d, Width * Width * sizeof(float));

  hipMemcpy(A_d, A_h, Width * Width * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, Width * Width * sizeof(float), hipMemcpyHostToDevice);

  dim3 dimGrid(Width / TILE_WIDTH, Width / TILE_WIDTH);
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  MatrixMulKernel<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, Width);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0.0;
  hipEventElapsedTime(&milliseconds, start, stop);

  std::cout << "Elapsed time for testMatrixMulKernel : " << milliseconds
            << '\n';

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  free(A_h);
  free(B_h);
}

int main(int argc, const char* argv[]) {
	hipDeviceSynchronize();
	std::cout << "Arithmetic Intensity : " << 1.0 / 3.0 * float(Width) << '\n';
  for(int i=0;i<1;i++){
    testMatrixMulKernel(Width);
  }
	return 0;
}