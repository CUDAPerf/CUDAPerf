
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

const int my_bitset_size = 512;
const int my_bunch_size = 1024*2;
typedef unsigned uint;

//using one thread per bitset in the bunch
__global__ void kernelXOR(uint * bitset, uint * bunch, int * set_bits, int bitset_size, int bunch_size) {

    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if (tid < bunch_size){      // 1 Thread for each bitset in the 'bunch'
        int sum = 0;
        uint xor_res = 0;
        for (int i = 0; i < bitset_size; ++i){  // Iterate through every uint-block of the bitsets
            xor_res = bitset[i] ^ bunch[bitset_size * tid + i];
            sum += __popc(xor_res);
        }
        set_bits[tid] = sum;
    }
}

__global__ void kernelXOR2(uint * bitset, uint * bunch, int * set_bits, int bitset_size, int bunch_size) {

    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if (tid < bunch_size){      // 1 Thread for each bitset in the 'bunch'
        int sum = 0;
        uint xor_res = 0;
        for (int i = 0; i < bitset_size; ++i){  // Iterate through every uint-block of the bitsets
            xor_res = bitset[i] ^ bunch[bitset_size * tid + i];
            sum += __popc(xor_res);
        }
        set_bits[tid] = sum;
    }
}

const int nTPB = 256;
// one block per bitset, multiple bitsets per block
__global__ void kernelXOR_imp(const uint * __restrict__  bitset, const uint * __restrict__  bunch, int * __restrict__  set_bits, int bitset_size, int bunch_size) {

    __shared__ uint sbitset[my_bitset_size];  // could also be dynamically allocated for varying bitset sizes
    __shared__ int ssum[nTPB];
    // load shared, block-stride loop
    for (int idx = threadIdx.x; idx < bitset_size; idx += blockDim.x) sbitset[idx] = bitset[idx];
    __syncthreads();
    // stride across all bitsets in bunch
    for (int bidx = blockIdx.x; bidx < bunch_size; bidx += gridDim.x){
      int my_sum = 0;
      for (int idx = threadIdx.x; idx < bitset_size; idx += blockDim.x) my_sum += __popc(sbitset[idx] ^ bunch[bidx*bitset_size + idx]);
    // block level parallel reduction
      ssum[threadIdx.x] = my_sum;
      for (int ridx = nTPB>>1; ridx > 0; ridx >>=1){
        __syncthreads();
        if (threadIdx.x < ridx) ssum[threadIdx.x] += ssum[threadIdx.x+ridx];}
      if (!threadIdx.x) set_bits[bidx] = ssum[0];}
}



int test(){

// data setup

  uint *d_cbitset, *d_bitsets, *h_cbitset, *h_bitsets;
  int *d_r, *h_r, *h_ri;
  h_cbitset = new uint[my_bitset_size];
  h_bitsets = new uint[my_bitset_size*my_bunch_size];
  h_r  = new int[my_bunch_size];
  h_ri = new int[my_bunch_size];
  for (int i = 0; i < my_bitset_size*my_bunch_size; i++){
    h_bitsets[i] = rand();
    if (i < my_bitset_size) h_cbitset[i] = rand();}
  hipMalloc(&d_cbitset, my_bitset_size*sizeof(uint));
  hipMalloc(&d_bitsets, my_bitset_size*my_bunch_size*sizeof(uint));
  hipMalloc(&d_r,  my_bunch_size*sizeof(int));
  hipMemcpy(d_cbitset, h_cbitset, my_bitset_size*sizeof(uint), hipMemcpyHostToDevice);
  hipMemcpy(d_bitsets, h_bitsets, my_bitset_size*my_bunch_size*sizeof(uint), hipMemcpyHostToDevice);

  // Grid/Blocks used for kernel invocation
  dim3 block(32);
  dim3 grid((my_bunch_size-1)/32+1);

  kernelXOR2<<<grid, block>>>(d_cbitset, d_bitsets, d_r, my_bitset_size, my_bunch_size);
  hipMemcpy(h_r, d_r, my_bunch_size*sizeof(int), hipMemcpyDeviceToHost);

  dim3 iblock(nTPB);
  dim3 igrid((my_bunch_size-1)/nTPB+1);

  kernelXOR<<<igrid, iblock>>>(d_cbitset, d_bitsets, d_r, my_bitset_size, my_bunch_size);
  hipMemcpy(h_r, d_r, my_bunch_size*sizeof(int), hipMemcpyDeviceToHost);


  kernelXOR_imp<<<igrid, iblock>>>(d_cbitset, d_bitsets, d_r, my_bitset_size, my_bunch_size);
  hipMemcpy(h_ri, d_r, my_bunch_size*sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < my_bunch_size; i++)
    if (h_r[i] != h_ri[i]) {std::cout << "mismatch at i: " << i << " was: " << h_ri[i] << " should be: " << h_r[i] << std::endl; return 0;}
  std::cout << "Results match." << std::endl;
  return 0;
}

int main(){
  hipSetDevice(1);
  for(int i = 0; i < 10; i++){
    test();
  }
  return 0;
}