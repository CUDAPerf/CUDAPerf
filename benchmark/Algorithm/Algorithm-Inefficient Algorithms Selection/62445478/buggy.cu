#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>
#include <cmath>

__global__ void modulus_kernel1(int length, int lines, hipfftComplex *PostFFTData, float* z) 
{
    //delete sync
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    if(x<length*lines)
        z[x] = sqrt(PostFFTData[x].x *PostFFTData[x].x + PostFFTData[x].y *PostFFTData[x].y);
}


void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "Error: " << msg << " (" << hipGetErrorString(err) << ")" << std::endl;
        exit(EXIT_FAILURE);
    }
}


int main() {
    hipSetDevice(1);
    int FFTlength = 1024; // Example FFT length
    int lines = 1024; // Example number of lines

    // Define block and grid sizes
    dim3 dimBlock(256);
    dim3 dimGrid((FFTlength * lines + dimBlock.x - 1) / dimBlock.x);

    // Allocate host memory
    hipfftComplex* h_PostFFTData = new hipfftComplex[FFTlength * lines];
    float* h_z = new float[FFTlength * lines];

    // Initialize host memory with some example values
    for (int i = 0; i < FFTlength * lines; ++i) {
        h_PostFFTData[i].x = static_cast<float>(i);
        h_PostFFTData[i].y = static_cast<float>(i) / 2.0f;
    }

    // Allocate device memory
    hipfftComplex* d_PostFFTData;
    float* d_z;
    checkCudaError(hipMalloc((void**)&d_PostFFTData, FFTlength * lines * sizeof(hipfftComplex)), "hipMalloc d_PostFFTData");
    checkCudaError(hipMalloc((void**)&d_z, FFTlength * lines * sizeof(float)), "hipMalloc d_z");

    // Copy data from host to device
    checkCudaError(hipMemcpy(d_PostFFTData, h_PostFFTData, FFTlength * lines * sizeof(hipfftComplex), hipMemcpyHostToDevice), "hipMemcpy h_PostFFTData to d_PostFFTData");
    // Launch kernel
    for(int i=0;i<10;i++){
        modulus_kernel1<<<dimGrid, dimBlock>>>(FFTlength, lines, d_PostFFTData, d_z);
    }
    checkCudaError(hipGetLastError(), "Kernel launch");

    // Copy results from device to host
    checkCudaError(hipMemcpy(h_z, d_z, FFTlength * lines * sizeof(float), hipMemcpyDeviceToHost), "hipMemcpy d_z to h_z");


    // Clean up
    delete[] h_PostFFTData;
    delete[] h_z;
    checkCudaError(hipFree(d_PostFFTData), "hipFree d_PostFFTData");
    checkCudaError(hipFree(d_z), "hipFree d_z");

    return 0;
}


