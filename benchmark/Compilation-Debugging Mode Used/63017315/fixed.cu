#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>   // or equivalently <cub/block/block_radix_sort.cuh>
#include <iostream>
const int ipt=8;
const int tpb=128;
__global__ void ExampleKernel(int *data)
{
    // Specialize BlockRadixSort for a 1D block of 128 threads owning 8 integer items each
    typedef hipcub::BlockRadixSort<int, tpb, ipt> BlockRadixSort;
    // Allocate shared memory for BlockRadixSort
    __shared__ typename BlockRadixSort::TempStorage temp_storage;
    // Obtain a segment of consecutive items that are blocked across threads
    int thread_keys[ipt];
    // just create some synthetic data in descending order 1023 1022 1021 1020 ...
    for (int i = 0; i < ipt; i++) thread_keys[i] = (tpb-1-threadIdx.x)*ipt+i;
    // Collectively sort the keys
    BlockRadixSort(temp_storage).Sort(thread_keys);
    __syncthreads();
    // write results to output array
    for (int i = 0; i < ipt; i++) data[blockIdx.x*ipt*tpb + threadIdx.x*ipt+i] = thread_keys[i];
}


int main(){

    const int blks = 1024;
    int *data;
    hipMalloc(&data, blks*ipt*tpb*sizeof(int));
    for(int i =0;i<10;i++)
        ExampleKernel<<<blks,tpb>>>(data);
    int *h_data = new int[blks*ipt*tpb];
    hipMemcpy(h_data, data, blks*ipt*tpb*sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; i++) std::cout << h_data[i] << " ";
    std::cout << std::endl;
}