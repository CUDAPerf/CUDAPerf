#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

#define N 1024*2  // Adjust the number of elements as needed
#define BLOCK_SIZE 256

// Compute1_1 use float
__global__ void Compute1_1(float2* p0, float2* p1, float2* v, int count) {
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    float2 fd = {0.0F, 0.0F};
    for (auto j = 0; j < count; ++j) {
        //if (i == j) continue;
        const auto dx = p0[i].x - p0[j].x;
        const auto dy = p0[i].y - p0[j].y;
        const auto f = 0.000000001F / (dx * dx + dy * dy + 0.000000000001F);
        fd.x += dx * f;
        fd.y += dy * f;
    }
    p1[i].x = p0[i].x + (v[i].x -= fd.x);
    p1[i].y = p0[i].y + (v[i].y -= fd.y);
}

int main() {
    hipSetDevice(1);
    int count = N;

    float2* oldPos, * newPos, * vel;
    float2* d_oldPos, * d_newPos, * d_vel;

    size_t size_float2 = count * sizeof(float2);

    oldPos = (float2*)malloc(size_float2);
    newPos = (float2*)malloc(size_float2);
    vel = (float2*)malloc(size_float2);

    hipMalloc(&d_oldPos, size_float2);
    hipMalloc(&d_newPos, size_float2);
    hipMalloc(&d_vel, size_float2);

    for (int i = 0; i < count; ++i) {      
        oldPos[i] = {i * 1.0f, i * 2.0f}; 
        vel[i] = {1.0f, 1.0f};         
    }

    hipMemcpy(d_oldPos, oldPos, size_float2, hipMemcpyHostToDevice);
    hipMemcpy(d_vel, vel, size_float2, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for(int i=0;i<10;i++){
        hipEventRecord(start);
        Compute1_1<<<(count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_oldPos, d_newPos, d_vel, count);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float elapsedTime1;
        hipEventElapsedTime(&elapsedTime1, start, stop);
        std::cout << "Compute1 execution time: " << elapsedTime1 << " ms" << std::endl;
    }

    hipFree(d_oldPos);
    hipFree(d_newPos);
    hipFree(d_vel);

    free(oldPos);
    free(newPos);
    free(vel);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}