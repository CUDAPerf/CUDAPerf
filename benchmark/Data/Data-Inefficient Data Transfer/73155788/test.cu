
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

const size_t mb20 = 1024 * 1024 * 512;


    hipError_t allocate(void** ptr, size_t size, hipStream_t stream){
        return hipMallocAsync(ptr, size, stream);
    }

    hipError_t deallocate(void* ptr, hipStream_t stream){
        return hipFreeAsync(ptr, stream);
    }


    hipError_t allocate2(void** ptr, size_t size, hipStream_t){
        return hipMalloc(ptr, size);
    }

    hipError_t deallocate2(void* ptr, hipStream_t){
        return hipFree(ptr);
    }


__global__
void computeNextSize(size_t* size, size_t growBy){
    *size = *size + growBy;
}

void method1(){
    hipStream_t stream = hipStreamPerThread;

    size_t* h_size = nullptr;
    size_t* d_size = nullptr;

    hipMalloc(&d_size, sizeof(size_t));
    hipHostMalloc(&h_size, sizeof(size_t), hipHostMallocDefault);
    *h_size = mb20;
    hipMemcpyAsync(d_size, h_size, sizeof(size_t), hipMemcpyHostToDevice, stream);

    hipMemPool_t memPool;
    hipDeviceGetMemPool(&memPool, 0);
    size_t setVal = UINT64_MAX;
    hipMemPoolSetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &setVal);

    void* ptr = nullptr;
    size_t size = mb20;
    allocate(&ptr, size, stream);
    //std::cout << "size: " << size << ", ptr = " << ptr << "\n";
    hipMemsetAsync(ptr, 0, size); //work with ptr
    computeNextSize<<<1,1,0,stream>>>(d_size, mb20);
    hipMemcpyAsync(h_size, d_size, sizeof(size_t), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream); //wait for computation and transfer of next size


    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for(int i = 0; i < 50; i++){
        deallocate(ptr, stream);
        size = *h_size;
        allocate(&ptr, size, stream);
        //std::cout << "size: " << size << ", ptr = " << ptr << "\n";
        hipMemsetAsync(ptr, 0, size); //work with ptr
        computeNextSize<<<1,1,0,stream>>>(d_size, mb20);
        hipMemcpyAsync(h_size, d_size, sizeof(size_t), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream); //wait for computation and transfer of next size
    }
    deallocate(ptr, stream);
    hipStreamSynchronize(stream);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "method1: " << elapsedTime << " ms" << std::endl;
}

void method2(){
    hipStream_t stream = hipStreamPerThread;

    size_t* h_size = nullptr;
    size_t* d_size = nullptr;

    hipMalloc(&d_size, sizeof(size_t));
    hipHostMalloc(&h_size, sizeof(size_t), hipHostMallocDefault);
    *h_size = mb20;
    hipMemcpyAsync(d_size, h_size, sizeof(size_t), hipMemcpyHostToDevice, stream);

    hipMemPool_t memPool;
    hipDeviceGetMemPool(&memPool, 0);
    size_t setVal = UINT64_MAX;
    hipMemPoolSetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &setVal);

    void* ptr = nullptr;
    size_t size = mb20;
    allocate2(&ptr, size, stream);
    //std::cout << "size: " << size << ", ptr = " << ptr << "\n";
    hipMemsetAsync(ptr, 0, size); //work with ptr
    computeNextSize<<<1,1,0,stream>>>(d_size, mb20);
    hipMemcpyAsync(h_size, d_size, sizeof(size_t), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream); //wait for computation and transfer of next size


    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for(int i = 0; i < 50; i++){
        deallocate2(ptr, stream);
        size = *h_size;
        allocate2(&ptr, size, stream);
        //std::cout << "size: " << size << ", ptr = " << ptr << "\n";
        hipMemsetAsync(ptr, 0, size); //work with ptr
        computeNextSize<<<1,1,0,stream>>>(d_size, mb20);
        hipMemcpyAsync(h_size, d_size, sizeof(size_t), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream); //wait for computation and transfer of next size
    }
    deallocate2(ptr, stream);
    hipStreamSynchronize(stream);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "method2: " << elapsedTime << " ms" << std::endl;
}

int main(){
    hipSetDevice(1);
    method1();
    method2();
}