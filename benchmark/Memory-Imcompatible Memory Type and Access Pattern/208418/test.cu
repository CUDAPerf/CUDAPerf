
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
// the movement
// start:
//  A B
//  C D
// step 1:
//  B A
//  C D
// step 2:
//  C A
//  B D
// step 3:
//  A C
//  B D

__global__ void t_bad(int do_print){

  int u[32];
  for (int i = 0; i < 32; i++) u[i] = threadIdx.x*32+i;
  if (u[0] >= do_print)
    // print data
    for (int i = 0; i < 32; i++)
      if (threadIdx.x == i){
        for (int j = 0; j < 32; j++)  printf("%d ", u[j]);
        printf("\n");}
  #pragma unroll 31
  for (int i = 1; i < 32; i++){
    int idx = threadIdx.x^i;
    u[idx] = __shfl_sync(0xFFFFFFFF, u[idx], idx);}
  if (u[0] >= do_print)
    // print data
    for (int i = 0; i < 32; i++)
      if (threadIdx.x == i){
        for (int j = 0; j < 32; j++)  printf("%d ", u[j]);
        printf("\n");}
}

template <typename T>
__device__ __forceinline__ void myswap(T &a, T &b){ T s = a;  a = b; b = s;}
template <typename T, int s>
__device__ __forceinline__ void mymove(T (&u)[32]){
  const int s1 = 2*s;
  // step 1:
  if (!(threadIdx.x&s)) {
    #pragma unroll 16
    for (int i = 0; i < 16; i++){
      int i1 = i%s;
      int i2 = i/s;
      int i3 = i2*s1;
      myswap(u[i3+i1], u[i3+i1+s]);}}
  // step 2:
  #pragma unroll 16
  for (int i = 0; i < 16; i++){
    int i1 = i%s;
    int i2 = i/s;
    int i3 = i2*s1;
    u[i3+i1] = __shfl_xor_sync(0xFFFFFFFF, u[i3+i1], s);}
  // step 3:
  if (!(threadIdx.x&s)) {
    #pragma unroll 16
    for (int i = 0; i < 16; i++){
      int i1 = i%s;
      int i2 = i/s;
      int i3 = i2*s1;
      myswap(u[i3+i1], u[i3+i1+s]);}}
}

template <typename T>
__global__ void t(int do_print){
  T u[32];
  // initialize data
  for (int i = 0; i < 32; i++)
    u[i] = threadIdx.x*32+i;
  if (u[0] > do_print)
    // print data
    for (int i = 0; i < 32; i++)
      if (threadIdx.x == i){
        for (int j = 0; j < 32; j++)  printf("%d ", u[j]);
        printf("\n");}
  mymove<T, 1>(u);
  mymove<T, 2>(u);
  mymove<T, 4>(u);
  mymove<T, 8>(u);
  mymove<T,16>(u);
  if (u[0] >= do_print)
    // print data
    for (int i = 0; i < 32; i++)
      if (threadIdx.x == i){
        for (int j = 0; j < 32; j++)  printf("%d ", u[j]);
        printf("\n");}
}

int main(int argc, char *argv[]){
  hipSetDevice(1);
  int do_print = 1024;
  if (argc > 1) do_print = atoi(argv[1]);
  int n = 1024*1024;
  for(int i=0;i<10;i++){
    t_bad<<<n,32>>>(do_print);
    hipDeviceSynchronize();
    t<int><<<n,32>>>(do_print);
    hipDeviceSynchronize();
  }
}