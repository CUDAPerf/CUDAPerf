#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

__global__ void setupCurandStates(hiprandState *states, unsigned long long seed, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

__global__ void dropr(float *A, hiprandState *globalstate, uint64_t N, float R) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        hiprandState localstate = globalstate[i];
        A[i] *= hiprand_uniform(&localstate) < R ? 0 : 1;
        globalstate[i] = localstate;
    }
}

void measureKernelTime(float *A, hiprandState *states, uint64_t N, float R) {
    float *d_A;
    hiprandState *d_states;

    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_states, N * sizeof(hiprandState));

    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);

    // Setup random states
    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    setupCurandStates<<<blocks, threadsPerBlock>>>(d_states, 1234, N);

    // Measure time for dropr
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    dropr<<<blocks, threadsPerBlock>>>(d_A, d_states, N, R);
    hipEventRecord(stop);
    
    hipEventSynchronize(stop);
    float time1;
    hipEventElapsedTime(&time1, start, stop);
    std::cout << "Time for dropr: " << time1 << " ms" << std::endl;

    // Reset states and measure time for dropr2
    hipMemcpy(d_states, d_states, N * sizeof(hiprandState), hipMemcpyDeviceToDevice);

    // Clean up
    hipFree(d_A);
    hipFree(d_states);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    //cudaSetDevice(1);
    const uint64_t N = 1 << 17; // Size of the array
    float R = 0.5f;
    float *A = new float[N];
    hiprandState *states = new hiprandState[N];

    // Initialize input data
    for (uint64_t i = 0; i < N; ++i) {
        A[i] = static_cast<float>(i) / N;
    }

    for(int i=0;i<10;i++){
        measureKernelTime(A, states, N, R);
    }


    delete[] A;
    delete[] states;
    return 0;
}