#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <vector>    
#include <chrono>
#include <algorithm>

// copy data directly
__global__ void copyRow(float * MatA,float * MatB,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x * 2;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    int idx=ix+iy*nx;

    if (ix<nx && iy<ny)
    {

      MatB[idx]  = MatA[idx];
      MatB[idx+blockDim.x] = MatA[idx+blockDim.x];
      
      
    }
}

__global__ void copyRow_reg(float * MatA,float * MatB,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x * 2;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    int idx=ix+iy*nx;

    if (ix<nx && iy<ny)
    {

      auto t1 = MatA[idx];
      auto t2 = MatA[idx+blockDim.x];
      MatB[idx] = t1;
      MatB[idx+blockDim.x]= t2;
    }
}

__global__ void copyRow_Sheme(float * MatA,float* MatB,int nx,int ny)
{
  __shared__ float tile[16][16 * 2];
  int ix=threadIdx.x+blockDim.x * blockIdx.x * 2;
  int iy=threadIdx.y+blockDim.y * blockIdx.y;
  int idx=ix+iy * nx;

   // loop unroll 2
   if(ix<nx&& iy<ny)
        {
          tile[threadIdx.y][threadIdx.x]=MatA[idx];
          tile[threadIdx.y][threadIdx.x+blockDim.x]=MatA[idx+blockDim.x];

          __syncthreads();

          MatB[idx]=tile[threadIdx.y][threadIdx.x];
          MatB[idx+blockDim.x]=tile[threadIdx.y][threadIdx.x+blockDim.x];
        }
}

using mt = float;
int main(){
  //cudaSetDevice(1);
  size_t sz = 1024*8;
  size_t msz = sz*sz;
  dim3 grid = dim3(sz/16/2, sz/16);
  dim3 block = dim3(16,16);
  mt *d_MatA, *d_MatB;
  hipMalloc(&d_MatA, sizeof(float)*msz);
  hipMalloc(&d_MatB, sizeof(float)*msz);
  for(int i=0;i<10;i++){
    copyRow<<<grid,block>>>(d_MatA, d_MatB, sz, sz);
    hipDeviceSynchronize();

    copyRow_Sheme<<<grid,block>>>(d_MatA,d_MatB,sz,sz);
    hipDeviceSynchronize();

    copyRow_reg<<<grid,block>>>(d_MatA, d_MatB, sz, sz);
    hipDeviceSynchronize();
  }
  
}
