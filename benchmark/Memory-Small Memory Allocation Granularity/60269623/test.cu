#include <iostream>
#include <hip/hip_runtime.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define N 102400
#define ARR_SZ 100

struct Struct
{
    float* arr;
};

int before()
{
    Struct* struct_arr;

    size_t free_mem, total_mem;
    hipError_t status = hipMemGetInfo(&free_mem, &total_mem);
    std::cout << "Used memory: " << (double)(total_mem - free_mem) / (1024 * 1024) << " MB" << std::endl;
    double mem1 = (double)(total_mem - free_mem);

    gpuErrchk( hipMallocManaged((void**)&struct_arr, sizeof(Struct)*N) );
    for(int i = 0; i < N; ++i)
        gpuErrchk( hipMallocManaged((void**)&(struct_arr[i].arr), sizeof(float)*ARR_SZ) ); //out of memory...

    status = hipMemGetInfo(&free_mem, &total_mem);
    std::cout << "Used memory: " << (double)(total_mem - free_mem) / (1024 * 1024) << " MB" << std::endl;
    double mem2 = (double)(total_mem - free_mem);
    std::cout<<"Memory used before: "<<(mem2-mem1)/1024<<"KB"<<std::endl;

    for(int i = 0; i < N; ++i)
        hipFree(struct_arr[i].arr);
    hipFree(struct_arr);

    /*float* f;
    gpuErrchk( cudaMallocManaged((void**)&f, sizeof(float)*N*ARR_SZ) ); //this works ok
    cudaFree(f);*/

    return 0;
}

int after(){
    Struct* struct_arr;
    float* f;

    size_t free_mem, total_mem;
    hipError_t status = hipMemGetInfo(&free_mem, &total_mem);
    std::cout << "Used memory: " << (double)(total_mem - free_mem) / (1024 * 1024) << " MB" << std::endl;
    double mem1 = (double)(total_mem - free_mem);
    gpuErrchk( hipMallocManaged((void**)&struct_arr, sizeof(Struct)*N) );
    gpuErrchk( hipMallocManaged((void**)&f, sizeof(float)*N*ARR_SZ) );
    for(int i = 0; i < N; ++i)
        struct_arr[i].arr = f+i*ARR_SZ;

    status = hipMemGetInfo(&free_mem, &total_mem);
    std::cout << "Used memory: " << (double)(total_mem - free_mem) / (1024 * 1024) << " MB" << std::endl;
    double mem2 = (double)(total_mem - free_mem);
    std::cout<<"Memory used after: "<<(mem2-mem1)/1024<<"KB"<<std::endl;

    hipFree(struct_arr);
    hipFree(f);

    return 0;
}

int main(){
    hipSetDevice(0);
    before();
    after();
}