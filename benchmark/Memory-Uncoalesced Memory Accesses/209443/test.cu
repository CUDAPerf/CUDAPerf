#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <vector>    
#include <chrono>
#include <algorithm>
#include <iostream>

#define N 8
#define MAX 1024*1024*512

__global__ void kernel1(int *x){
    int idx = blockDim.x* blockIdx.x + threadIdx.x;
    #pragma unroll
    for(auto i =0; i<N; i++){
        int j = idx*N + i;
        if(j>MAX) break;
        x[j] = j; /* do something with x[j] */
    }
}

__global__ void kernel2(int*x){
    #pragma unroll
    for(auto i = 0; i < N; i++) {
        int idx = blockDim.x * blockIdx.x + threadIdx.x;
        int j = i * gridDim.x * blockDim.x + idx; 
        if(j > MAX) break;
        x[j] = j; /* do something with x[j] */
    }
}

void checkResults(int *x1, int *x2, int size) {
    for (int i = 0; i < size; i++) {
        if (x1[i] != x2[i]) {
            std::cout << "Mismatch at index " << i << ": " << x1[i] << " != " << x2[i] << std::endl;
            return;
        }
    }
    std::cout << "Results match!" << std::endl;
}

int main() {
    hipSetDevice(1);
    int Q = MAX / (N * 512);
    int *d_x1, *d_x2;
    int *h_x1 = new int[MAX];
    int *h_x2 = new int[MAX];

    hipMalloc(&d_x1, sizeof(int) * MAX);
    hipMalloc(&d_x2, sizeof(int) * MAX);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Run kernel1
    hipEventRecord(start);
    for(int i=0;i<10;i++){
        kernel1<<<Q, 512>>>(d_x1);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time1;
    hipEventElapsedTime(&time1, start, stop);
    std::cout << "Kernel1 time: " << time1 << " ms" << std::endl;

    // Run kernel2
    hipEventRecord(start);
    for(int i=0;i<10;i++){
        kernel2<<<Q, 512>>>(d_x2);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time2;
    hipEventElapsedTime(&time2, start, stop);
    std::cout << "Kernel2 time: " << time2 << " ms" << std::endl;

    // Copy results back
    hipMemcpy(h_x1, d_x1, sizeof(int) * MAX, hipMemcpyDeviceToHost);
    hipMemcpy(h_x2, d_x2, sizeof(int) * MAX, hipMemcpyDeviceToHost);

    // Check results
    checkResults(h_x1, h_x2, MAX);

    // Clean up
    delete[] h_x1;
    delete[] h_x2;
    hipFree(d_x1);
    hipFree(d_x2);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}