#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <vector>    
#include <chrono>
#include <algorithm>


__global__ void copyRow_reg(float * MatA,float * MatB,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x * 2;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    int idx=ix+iy*nx;

    if (ix<nx && iy<ny)
    {

      auto t1 = MatA[idx];
      auto t2 = MatA[idx+blockDim.x];
      MatB[idx] = t1;
      MatB[idx+blockDim.x]= t2;
    }
}

using mt = float;
int main(){
  //cudaSetDevice(1);
  size_t sz = 1024*8;
  size_t msz = sz*sz;
  dim3 grid = dim3(sz/16/2, sz/16);
  dim3 block = dim3(16,16);
  mt *d_MatA, *d_MatB;
  hipMalloc(&d_MatA, sizeof(float)*msz);
  hipMalloc(&d_MatB, sizeof(float)*msz);
  for(int i=0;i<10;i++){
    copyRow_reg<<<grid,block>>>(d_MatA, d_MatB, sz, sz);
    hipDeviceSynchronize();
  }
  
}
