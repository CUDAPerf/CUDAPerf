#include <hip/hip_runtime.h>

const int rows = 1024*64;
const int columns = 1024*8;
const int tile_dim = 32;

__global__ void transpose(float* in, float* out)
{
  __shared__ float tile[tile_dim][tile_dim + 1];

  int x = blockIdx.x * tile_dim + threadIdx.x;
  int y = blockIdx.y * tile_dim + threadIdx.y;

  tile[threadIdx.y][threadIdx.x] = in[y * columns + x];

  __syncthreads();

  x = blockIdx.y * tile_dim + threadIdx.x;
  y = blockIdx.x * tile_dim + threadIdx.y;

  out[y * columns + x] = tile[threadIdx.x][threadIdx.y];
}

int main()
{
  //cudaSetDevice(1);
  float *in, *out;

  size_t size = rows * columns * sizeof(float);
  hipMalloc(&in, size);
  hipMalloc(&out, size);

  dim3 grid(rows / tile_dim, columns / tile_dim);
  dim3 block(tile_dim, tile_dim);
  for(int i =0;i<10;i++){
    transpose<<<grid, block>>>(in, out);
  }

  hipDeviceSynchronize();

  return 0;
}