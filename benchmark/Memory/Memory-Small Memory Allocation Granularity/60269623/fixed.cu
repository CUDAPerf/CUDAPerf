#include <iostream>
#include <hip/hip_runtime.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define N 102400
#define ARR_SZ 100

struct Struct
{
    float* arr;
};

int after(){
    Struct* struct_arr;
    float* f;

    size_t free_mem, total_mem;
    hipError_t status = hipMemGetInfo(&free_mem, &total_mem);
    std::cout << "Used memory: " << (double)(total_mem - free_mem) / (1024 * 1024) << " MB" << std::endl;
    double mem1 = (double)(total_mem - free_mem);
    gpuErrchk( hipMallocManaged((void**)&struct_arr, sizeof(Struct)*N) );
    gpuErrchk( hipMallocManaged((void**)&f, sizeof(float)*N*ARR_SZ) );
    for(int i = 0; i < N; ++i)
        struct_arr[i].arr = f+i*ARR_SZ;

    status = hipMemGetInfo(&free_mem, &total_mem);
    std::cout << "Used memory: " << (double)(total_mem - free_mem) / (1024 * 1024) << " MB" << std::endl;
    double mem2 = (double)(total_mem - free_mem);
    std::cout<<"Memory used after: "<<(mem2-mem1)/1024<<"KB"<<std::endl;

    hipFree(struct_arr);
    hipFree(f);

    return 0;
}

int main(){
    hipSetDevice(0);
    after();
}