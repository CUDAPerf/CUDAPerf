#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <algorithm>
#include <cmath>
#include <iostream>
#include <vector>
#include <ctime>
#include <ratio>
#include <chrono>
#include <random>
#include <time.h>
#include <math.h>

// CUDA
#include <hip/hip_runtime.h>


#include <hipsolver.h>

//#include "Utilities.cuh"

using namespace std;
using namespace std::chrono;

/************************************/
/* COEFFICIENT REARRANGING FUNCTION */
/************************************/
void rearrange(double** vec, int* pivotArray, int N, int numMatrices) {
  for (int nm = 0; nm < numMatrices; nm++) {
    for (int i = 0; i < N; i++) {
      double temp = vec[nm][i];
      vec[nm][i] = vec[nm][pivotArray[N*i + nm] - 1];
      vec[nm][pivotArray[N * i + nm] - 1] = temp;
    }
  }
}


/************************************/
/* MAIN  */
/************************************/
int main() {
  hipSetDevice(1);

  const int N = 3; 
  const int numMatrices = 1024*1024*512; 

  // random generator to fill matrices and coefficients
  random_device device;
  mt19937 generator(device());
  uniform_real_distribution<double> distribution(1., 5.);
  //ALLOCATE MEMORY - using unified memory
  double** h_A;
  hipMallocManaged(&h_A, sizeof(double*) * numMatrices);
  for (int nm = 0; nm < numMatrices; nm++) {
    hipMallocManaged(&(h_A[nm]), sizeof(double) * N * N);
  }

  double** h_b;
  hipMallocManaged(&h_b, sizeof(double*) * numMatrices);
  for (int nm = 0; nm < numMatrices; nm++) {
    hipMallocManaged(&(h_b[nm]), sizeof(double) * N );
  }
  cout << " memory allocated" << endl;
  // FILL MATRICES
  for (int nm = 0; nm < numMatrices; nm++) {
    for (int i = 0; i < N; i++) {
      for (int j = 0; j < N; j++) {
        h_A[nm][j * N + i] = distribution(generator);
      }
    }
  }
  cout << " Matrix filled " << endl;

  // FILL COEFFICIENTS
  for (int nm = 0; nm < numMatrices; nm++) {
    for (int i = 0; i < N; i++) {
      h_b[nm][i] = distribution(generator);
    }
  }
  cout << " Coeff. vector filled " << endl;
  cout << endl;

  // --- CUDA solver initialization
  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);
  int* PivotArray;
  hipMallocManaged(&PivotArray, N * numMatrices * sizeof(int));
  int* infoArray;
  hipMallocManaged(&infoArray, numMatrices * sizeof(int));

  //CUBLAS LU SOLVER
  high_resolution_clock::time_point t1 = high_resolution_clock::now();
  hipblasDgetrfBatched(cublas_handle, N, h_A, N, PivotArray, infoArray, numMatrices);
  hipDeviceSynchronize();
  high_resolution_clock::time_point t2 = high_resolution_clock::now();
  duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
  cout << "It took " << time_span.count() * 1000. << " milliseconds." << endl;


  for (int i = 0; i < numMatrices; i++)
    if (infoArray[i] != 0) {
      fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", i);
    }

 // rearrange coefficient 
 // (temporarily on CPU, this step will be on a GPU Kernel as well)
  high_resolution_clock::time_point tA = high_resolution_clock::now();
  rearrange(h_b, PivotArray, N, numMatrices);
  high_resolution_clock::time_point tB = high_resolution_clock::now();
  duration<double> time_spanA = duration_cast<duration<double>>(tB - tA);
  cout << "rearrangement took " << time_spanA.count() * 1000. << " milliseconds." << endl;

//INVERT UPPER AND LOWER TRIANGULAR MATRICES 
  // --- Function solves the triangular linear system with multiple right-hand sides
  // --- Function overrides b as a result 
  const double alpha = 1.f;
  high_resolution_clock::time_point t3 = high_resolution_clock::now();
  hipblasDtrsmBatched(cublas_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, N, 1, &alpha, h_A, N, h_b, N, numMatrices);
  hipblasDtrsmBatched(cublas_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, N, 1, &alpha, h_A, N, h_b, N, numMatrices);
  hipDeviceSynchronize();
  high_resolution_clock::time_point t4 = high_resolution_clock::now();
  duration<double> time_span2 = duration_cast<duration<double>>(t4 - t3);
  cout << "second step took " << time_span2.count() * 1000. << " milliseconds." << endl;
  
  // --- Free resources
  if (h_A) hipFree(h_A);
  if (h_b) hipFree(h_b);
 
  hipDeviceReset();

  return 0;
}