#include "hip/hip_runtime.h"
#include <stdio.h>

void pascal(int *p, int n) {
	n--;
	p[0] = 1;
	for(int k = 0; k < (n >> 1); k++) {
		p[k + 1] = p[k] * (n - k) / (k + 1);
	}
}

__global__
void kernel1(unsigned short *__restrict__ result,const unsigned short *__restrict__ img, int width, int height, size_t result_pitch, size_t img_pitch, int n, const int *__restrict__ filter) {
	int i, j, z, k, l, c, m;
	z = blockIdx.x;
	i = blockIdx.y * blockDim.y + threadIdx.y;
	j = blockIdx.z * blockDim.z + threadIdx.z;
	if(i < width && j < height) {
		c = 0;
		for(k = 0; k < n >> 1; k++) {
			l = i + k - n / 2;
			m = 0;
			if(0 <= l && l < width) {
				m = img[(z * height + j) * img_pitch + l];
			}
			l = i + n - 1 - k - n / 2;
			if(0 <= l && l < width) {
				m += img[(z * height + j) * img_pitch + l];
			}
			c += filter[k] * m;
		}
		l = i + k - n / 2;
		if(0 <= l && l < width) {
			c += filter[k] * img[(z * height + j) * img_pitch + l];
		}
		result[(z * height + j) * result_pitch + i] = c >> (n - 1);
	}
}

__global__
void kernel2(unsigned short *__restrict__ result, const unsigned short *__restrict__ img, int width, int height, size_t result_pitch, size_t img_pitch, int n, const int *__restrict__ filter) {
	int i, j, z, k, l, c, m;
	z = blockIdx.x;
	i = blockIdx.y * blockDim.y + threadIdx.y;
	j = blockIdx.z * blockDim.z + threadIdx.z;
	if(i < width && j < height) {
		c = 0;
		for(k = 0; k < n >> 1; k++) {
			m = 0;
			l = i + k - n / 2;
			if(0 <= l && l < width) {
				m = img[(j * img_pitch + l * 3) + z];
			}
			l = i + n - 1 - k - n / 2;
			if(0 <= l && l < width) {
				m += img[(j * img_pitch + l * 3) + z];
			}
			c += filter[k] * m;
		}
		l = i + k - n / 2;
		if(0 <= l && l < width) {
			c += filter[k] * img[(j * img_pitch + l * 3) + z];
		}
		result[(j * result_pitch + i * 3) + z] = c >> (n - 1);
	}
}

void blur1(int width, int height) {
	int i, *__restrict__ filter, *__restrict__ filter_d;
	size_t img1_pitch, img2_pitch;
	unsigned short *__restrict__ img1, *__restrict__ img2;
	dim3 blocks(3, (width + 31) / 32, (height + 31) / 32);
	dim3 threadsPerBlock(1, 32, 32);

	filter = (int *)malloc(sizeof(int) * 9);
	pascal(filter, 17);
	hipMalloc((void **)&filter_d, sizeof(int) * 9);
	hipMemcpy(filter_d, filter, sizeof(int) * 9, hipMemcpyHostToDevice);

	hipMallocPitch((void **)&img1, &img1_pitch, sizeof(unsigned short) * width, height * 3);
	img1_pitch /= sizeof(unsigned short);
	hipMallocPitch((void **)&img2, &img2_pitch, sizeof(unsigned short) * width, height * 3);
	img2_pitch /= sizeof(unsigned short);
	for(i = 0; i < 10; i++) {
		// CALL kerne1 OR kernel2
		kernel1 << <blocks, threadsPerBlock >> > (img2, img1, width, height, img2_pitch, img1_pitch, 17, filter_d);
	}
	hipFree(img1);
	hipFree(img2);
	hipFree(filter_d);
	free(filter);
	hipDeviceSynchronize();
}

void blur2(int width, int height) {
	int i, *__restrict__ filter, *__restrict__ filter_d;
	size_t img1_pitch, img2_pitch;
	unsigned short *__restrict__ img1, *__restrict__ img2;
	dim3 blocks(3, (width + 31) / 32, (height + 31) / 32);
	dim3 threadsPerBlock(1, 32, 32);

	filter = (int *)malloc(sizeof(int) * 9);
	pascal(filter, 17);
	hipMalloc((void **)&filter_d, sizeof(int) * 9);
	hipMemcpy(filter_d, filter, sizeof(int) * 9, hipMemcpyHostToDevice);

	hipMallocPitch((void **)&img1, &img1_pitch, sizeof(unsigned short) * width, height * 3);
	img1_pitch /= sizeof(unsigned short);
	hipMallocPitch((void **)&img2, &img2_pitch, sizeof(unsigned short) * width, height * 3);
	img2_pitch /= sizeof(unsigned short);
	for(i = 0; i < 10; i++) {
		// CALL kerne1 OR kernel2
		kernel2 << <blocks, threadsPerBlock >> > (img2, img1, width, height, img2_pitch, img1_pitch, 17, filter_d);
	}
	hipFree(img1);
	hipFree(img2);
	hipFree(filter_d);
	free(filter);
	hipDeviceSynchronize();
}

int main(void) {
	hipSetDevice(1);
	clock_t begin, end;
	int width = 4096*8;
	int height = 4096*8;
	begin = clock();
	blur1(width, height);
	end = clock();
	printf("Time1: %f\n", (double)(end - begin) / CLOCKS_PER_SEC);
    begin = clock();
	blur2(width, height);
	end = clock();
	printf("Time2: %f\n", (double)(end - begin) / CLOCKS_PER_SEC);
	return 0;
}