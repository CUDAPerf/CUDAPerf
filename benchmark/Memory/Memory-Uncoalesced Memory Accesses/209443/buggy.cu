#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <vector>    
#include <chrono>
#include <algorithm>
#include <iostream>

#define N 8
#define MAX 1024*1024*512

__global__ void kernel1(int *x){
    int idx = blockDim.x* blockIdx.x + threadIdx.x;
    #pragma unroll
    for(auto i =0; i<N; i++){
        int j = idx*N + i;
        if(j>MAX) break;
        x[j] = j; /* do something with x[j] */
    }
}


int main() {
    hipSetDevice(1);
    int Q = MAX / (N * 512);
    int *d_x1;
    int *h_x1 = new int[MAX];

    hipMalloc(&d_x1, sizeof(int) * MAX);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Run kernel1
    hipEventRecord(start);
    for(int i=0;i<10;i++){
        kernel1<<<Q, 512>>>(d_x1);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time1;
    hipEventElapsedTime(&time1, start, stop);
    std::cout << "Kernel1 time: " << time1 << " ms" << std::endl;

    // Copy results back
    hipMemcpy(h_x1, d_x1, sizeof(int) * MAX, hipMemcpyDeviceToHost);


    // Clean up
    delete[] h_x1;
    hipFree(d_x1);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}