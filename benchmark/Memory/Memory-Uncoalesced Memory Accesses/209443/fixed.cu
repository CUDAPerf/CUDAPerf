#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <vector>    
#include <chrono>
#include <algorithm>
#include <iostream>

#define N 8
#define MAX 1024*1024*512

__global__ void kernel2(int*x){
    #pragma unroll
    for(auto i = 0; i < N; i++) {
        int idx = blockDim.x * blockIdx.x + threadIdx.x;
        int j = i * gridDim.x * blockDim.x + idx; 
        if(j > MAX) break;
        x[j] = j; /* do something with x[j] */
    }
}


int main() {
    hipSetDevice(1);
    int Q = MAX / (N * 512);
    int *d_x2;
    int *h_x2 = new int[MAX];

    hipMalloc(&d_x2, sizeof(int) * MAX);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Run kernel2
    hipEventRecord(start);
    for(int i=0;i<10;i++){
        kernel2<<<Q, 512>>>(d_x2);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time2;
    hipEventElapsedTime(&time2, start, stop);
    std::cout << "Kernel2 time: " << time2 << " ms" << std::endl;

    // Copy results back
    hipMemcpy(h_x2, d_x2, sizeof(int) * MAX, hipMemcpyDeviceToHost);

    // Clean up
    delete[] h_x2;
    hipFree(d_x2);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}