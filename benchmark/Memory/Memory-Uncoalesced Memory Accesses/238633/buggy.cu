#include <iostream>
#include <hip/hip_runtime.h>

#define N 1024*8

__global__ void kernel2(float* ma, float* mb, float* mc)
{
    uint32_t const row{blockIdx.x * blockDim.x + threadIdx.x};
    uint32_t const col{blockIdx.y * blockDim.y + threadIdx.y};

    uint32_t offset{N * row};
    float result{0.0f};

    for (uint32_t s{0}; s < N; ++s)
    {
        result += ma[offset + s] * mb[col + s * N];
    }

    mc[offset + col] = result;
}


__global__ void transpose(float* src, float* dest)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < N && y < N)
    {
        dest[y * N + x] = src[x * N + y];
    }
}


int main()
{
    hipSetDevice(1);
    
    float *h_a = new float[N * N];
    float *h_b = new float[N * N];
    float *h_c2 = new float[N * N];

    for (int i = 0; i < N * N; ++i)
    {
        h_a[i] = 1.0f; 
        h_b[i] = 2.0f; 
    }

    
    float *d_a, *d_b, *d_c2, *d_b_transpose, *d_a_transpose;
    hipMalloc(&d_a, N * N * sizeof(float));
    hipMalloc(&d_b, N * N * sizeof(float));
    hipMalloc(&d_c2, N * N * sizeof(float));
    hipMalloc(&d_b_transpose, N * N * sizeof(float));
    hipMalloc(&d_a_transpose, N * N * sizeof(float));

    
    hipMemcpy(d_a, h_a, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * N * sizeof(float), hipMemcpyHostToDevice);

    
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    transpose<<<numBlocks, threadsPerBlock>>>(d_b, d_b_transpose);
    transpose<<<numBlocks, threadsPerBlock>>>(d_a, d_a_transpose);
    hipDeviceSynchronize();

    hipEventRecord(start); 
    for(int i=0;i<10;i++){
        kernel2<<<numBlocks, threadsPerBlock>>>(d_a_transpose, d_b_transpose, d_c2);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop); 
    hipEventSynchronize(stop); 
    float elapsedTime2;
    hipEventElapsedTime(&elapsedTime2, start, stop); 
    std::cout << "Kernel2 execution time: " << elapsedTime2 << " ms\n";

    
    hipMemcpy(h_c2, d_c2, N * N * sizeof(float), hipMemcpyDeviceToHost);

    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c2);
    hipFree(d_b_transpose);
    hipFree(d_a_transpose);
    delete[] h_a;
    delete[] h_b;
    delete[] h_c2;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
