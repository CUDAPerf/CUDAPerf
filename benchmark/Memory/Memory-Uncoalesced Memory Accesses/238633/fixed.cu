#include <iostream>
#include <hip/hip_runtime.h>

#define N 1024*8

__global__ void kernel1(float* ma, float* mb, float* mc)
{
    uint32_t const row{blockIdx.y * blockDim.y + threadIdx.y};
    uint32_t const col{blockIdx.x * blockDim.x + threadIdx.x};

    uint32_t offset{N * row};
    float result{0.0f};

    for (uint32_t s{0}; s < N; ++s)
    {
        result += ma[offset + s] * mb[col + s * N];
    }

    mc[offset + col] = result;
}


int main()
{
    hipSetDevice(1);
    
    float *h_a = new float[N * N];
    float *h_b = new float[N * N];
    float *h_c1 = new float[N * N];

    for (int i = 0; i < N * N; ++i)
    {
        h_a[i] = 1.0f; 
        h_b[i] = 2.0f; 
    }

    
    float *d_a, *d_b, *d_c1, *d_b_transpose, *d_a_transpose;
    hipMalloc(&d_a, N * N * sizeof(float));
    hipMalloc(&d_b, N * N * sizeof(float));
    hipMalloc(&d_c1, N * N * sizeof(float));
    hipMalloc(&d_b_transpose, N * N * sizeof(float));
    hipMalloc(&d_a_transpose, N * N * sizeof(float));

    
    hipMemcpy(d_a, h_a, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * N * sizeof(float), hipMemcpyHostToDevice);

    
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); 
    for(int i=0;i<10;i++){
        kernel1<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c1);
        hipDeviceSynchronize();
    }
    hipEventRecord(stop); 
    hipEventSynchronize(stop); 
    float elapsedTime1;
    hipEventElapsedTime(&elapsedTime1, start, stop); 
    std::cout << "Kernel1 execution time: " << elapsedTime1 << " ms\n";

    
    hipMemcpy(h_c1, d_c1, N * N * sizeof(float), hipMemcpyDeviceToHost);

    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c1);
    hipFree(d_b_transpose);
    hipFree(d_a_transpose);
    delete[] h_a;
    delete[] h_b;
    delete[] h_c1;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
