#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdint.h>
#include <stdio.h>
#include <sys/time.h>

const int imageWidth = 1280;
const int imageHeight = 720;

// The "image" data pY is split into rectangles of the following dimension, with each rectangle processed by its own CUDA thread.
// There are 320 rectangles width-wise and 6 rectangles height-wise to perfectly cover the image area.
const int threadWidth = 4;
const int threadHeight = 120;

#define SQUARE(A)       ((A) * (A))

uint64_t get_time_usec(void)
{
	static struct timeval _time_stamp;
	gettimeofday(&_time_stamp, NULL);
	return (_time_stamp.tv_sec * 1000000ll) + _time_stamp.tv_usec;
}

__global__ void kernel3(uint8_t* pY)
{
    int cudaRow = blockIdx.y * blockDim.y + threadIdx.y;
    int pixelRow = cudaRow * threadHeight;
    int cudaCol = blockIdx.x * blockDim.x + threadIdx.x;
    int pixelCol = cudaCol * threadWidth;
    
    int xConst = (imageWidth / 2) - pixelCol;
    int yConst = (imageHeight / 2) - pixelRow;

    //assuming threadWidth == 4;

    for(int y_offset = 0; y_offset < threadHeight; y_offset++)
    {
        uint32_t offset = (pixelRow + y_offset) * imageWidth + pixelCol;
        uint8_t myPY[4];
        *((char4*)(&myPY[0])) = *((const char4*)(&pY[offset]));

        for(int x = 0; x < 4; x++){
          int xSq = SQUARE(xConst - x);
          int ySq = SQUARE(yConst - y_offset);
          int sumSq = xSq + ySq;
          float distanceFromCentre = sqrtf(sumSq);
          float correction = (1.0749947E-6f * sumSq) - (0.000297173f * distanceFromCentre) + 1.01820957f;
          float pixelVal = (float)myPY[x] * correction;
          myPY[x] = (uint8_t)pixelVal;
        }
        *((char4*)(&pY[offset])) = *((const char4*)(&myPY[0]));
        
        
    }
}


int main(void)
{
    //cudaSetDevice(1);
    //dim3 blocks = 2; // max on Jetson Nano
    dim3 threadsPerBlock = dim3(160, 6);
    int blockWidth = threadWidth * 160;
    int blockHeight = threadHeight * 6;
    dim3 blocks(
        (imageWidth + blockWidth - 1) / blockWidth, 
        (imageHeight + blockHeight - 1) / blockHeight
    );
    uint8_t* pY;
    
    hipMalloc(&pY, imageWidth * imageHeight * sizeof(uint8_t));
    
    uint64_t start = get_time_usec();
    
    for(int i=0;i<10;i++){
        kernel3<<<blocks, threadsPerBlock>>>(pY);
        hipDeviceSynchronize();
    }
    
     printf("kernel3 took %fms.\n", (double)(get_time_usec() - start) / 1000.0);
     
     return 0;
}