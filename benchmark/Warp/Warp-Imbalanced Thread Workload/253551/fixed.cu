#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <vector>    
#include <chrono>
#include <algorithm>

#define TILE_W 16
#define TILE_H 16
#define BLOCK 512

inline int get_number_of_blocks(int array_size, int block_size)
{
    return array_size / block_size + ((array_size % block_size > 0) ? 1 : 0);
}

__device__ __constant__ float kernel_gauss5x5[25] =
{
    0.00296902,      0.0133062,       0.0219382,       0.0133062,       0.00296902,
    0.0133062,       0.0596343,       0.0983203,       0.0596343,       0.0133062,
    0.0219382,       0.0983203,       0.162103,        0.0983203,       0.0219382,
    0.0133062,       0.0596343,       0.0983203,       0.0596343,       0.0133062,
    0.00296902,      0.0133062,       0.0219382,       0.0133062,       0.00296902
};

__device__ __constant__ float kernel_gauss7x7[49] =
{
    0.00001965,	0.00023941,	0.00107296,	0.00176901,	0.00107296,	0.00023941,	0.00001965,
    0.00023941,	0.0029166,	0.01307131,	0.02155094,	0.01307131,	0.0029166,	0.00023941,
    0.00107296,	0.01307131,	0.05858154,	0.09658462,	0.05858154,	0.01307131,	0.00107296,
    0.00176901,	0.02155094,	0.09658462,	0.15924113,	0.09658462,	0.02155094,	0.00176901,
    0.00107296,	0.01307131,	0.05858154,	0.09658462,	0.05858154,	0.01307131,	0.00107296,
    0.00023941,	0.0029166,	0.01307131,	0.02155094,	0.01307131,	0.0029166,	0.00023941,
    0.00001965,	0.00023941,	0.00107296,	0.00176901,	0.00107296,	0.00023941,	0.00001965,
};


__global__ void gauss5x5_tiles_kernel(const float* __restrict__ in, float *out, int w, int h)
{   
    const int R = 2;
    const int BLOCK_W = (TILE_W + 2*R);
    const int BLOCK_H = (TILE_H + 2*R);
    __shared__ float smem[BLOCK_W*BLOCK_H];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int offset = blockIdx.x * TILE_W + tx-R;
    int x = offset % w;
    int c = offset / w;
    int y = blockIdx.y * TILE_H + ty-R;

    // clamp to edge of image
    x = max(0, x);
    x = min(x, w-1);
    y = max(y, 0);
    y = min(y, h-1);
    //x = clamp(x, 0, w - 1);
    //y = clamp(y, 0, h - 1);

    unsigned int idx = y*w*4 + c*w + x;
    unsigned int bindex = threadIdx.y*BLOCK_W+threadIdx.x;
    
    // each thread copies its pixel of the block to shared memory
    smem[bindex] = in[idx];
    __syncthreads();

    float sum = 0;

    // only threads inside the apron will write results
    if (threadIdx.x >= R && threadIdx.x < (BLOCK_W-R) && threadIdx.y >= R && threadIdx.y < (BLOCK_H-R))
    {
#pragma unroll
        for(int i = -R; i <=R; i++)
        for(int j = -R; j <=R; j++)
        {
            sum += smem[bindex + (i*blockDim.x) + j] * kernel_gauss5x5[(i + R) * 5 + (j + R)];
        }

        out[idx] = sum;
    }
}

__global__ void gauss7x7_tiles_kernel(const float* __restrict__ in, float *out, int w, int h)
{   
    const int R = 3;
    const int BLOCK_W = (TILE_W + 2*R);
    const int BLOCK_H = (TILE_H + 2*R);
    __shared__ float smem[BLOCK_W*BLOCK_H];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int offset = blockIdx.x * TILE_W + tx-R;
    unsigned int x = offset % w;
    unsigned int c = offset / w;
    int y = blockIdx.y * TILE_H + ty-R;

    // clamp to edge of image
    x = max(0, x);
    x = min(x, w-1);
    y = max(y, 0);
    y = min(y, h-1);

    unsigned int idx = y*w*4 + c*w + x;
    unsigned int bindex = threadIdx.y*BLOCK_W+threadIdx.x;
    
    // each thread copies its pixel of the block to shared memory
    smem[bindex] = in[idx];
    __syncthreads();

    float sum = 0;

    // only threads inside the apron will write results
    if (threadIdx.x >= R && threadIdx.x < (BLOCK_W-R) && threadIdx.y >= R && threadIdx.y < (BLOCK_H-R))
    {
#pragma unroll
        for(int i = -R; i <=R; i++)
        for(int j = -R; j <=R; j++)
        {
            sum += smem[bindex + (i*blockDim.x) + j] *kernel_gauss7x7[(i + R) * 7 + (j + R)];
        }

        out[idx] = sum;
    }
}

extern "C" float* gauss5x5_gpu_tiles(float* d_src, float* d_dest, int w, int h, int cycles)
{
    dim3 dimGrid ((w*3) / TILE_W, h / TILE_H);
    dim3 dimBlock(TILE_W+4, TILE_H+4);

    float* src = d_src,*dst = d_dest, *tmp = d_dest;
    while (cycles--)
    {
        gauss5x5_tiles_kernel << < dimGrid, dimBlock, 0 >> > (src, d_dest, w, h);
        tmp = dst;
        dst = src;
        src = tmp;
    }

    return tmp;
}


extern "C" float* gauss7x7_gpu_tiles(float* d_src, float* d_dest, int w, int h, int cycles)
{
    dim3 dimGrid ((w*3) / TILE_W, h / TILE_H);
    dim3 dimBlock(TILE_W+6, TILE_H+6);

    float* src = d_src,*dst = d_dest, *tmp = d_dest;
    while (cycles--)
    {
        gauss7x7_tiles_kernel << < dimGrid, dimBlock, 0 >> > (src, d_dest, w, h);
        tmp = dst;
        dst = src;
        src = tmp;
    }

    return tmp;
}


void check_error(hipError_t status)
{
    hipError_t status2 = hipGetLastError();
    if (status != hipSuccess)
    {
        const char *s = hipGetErrorString(status);
        printf("\n CUDA Error: %s\n", s);
        getchar();
    }
    if (status2 != hipSuccess)
    {
        const char *s = hipGetErrorString(status2);
        printf("\n CUDA Error Prev: %s\n", s);
        getchar();
    }
}

void printTime(const char* name, double time)
{
    float fps = 1000 / time;
    printf("%-#40s",name);
    char tmp[32];
    sprintf(tmp, "%0.2f [ms]", time);
    printf("%-#20s%0.2f\n", tmp, fps);
}

#define CHECK_CUDA(X) check_error((hipError_t)X);

extern "C" float* gauss5x5_gpu_tiles(float* d_src, float* d_dest,  int w, int h, int cycles);
extern "C" float* gauss7x7_gpu_tiles(float* d_src, float* d_dest, int w, int h, int cycles);

int main(void)
{
    hipSetDevice(1);
    const int IMAGE_W = 2048*4 ; // pixels
    const int IMAGE_H = 2048*4 ;   
    const int N = IMAGE_W * IMAGE_H * 4;
    const int cycles = 10;

    // image is loaded as RGBA. fill with random values
    float* img_cpu = new float[N];
    for (int k = 0; k < N; k++)
        img_cpu[k] = std::rand() % 255;
  
    float* img_gpu = nullptr;
    CHECK_CUDA(hipMalloc((void **) &img_gpu, (N * sizeof(float))));

    float* temp_gpu = nullptr;
    CHECK_CUDA(hipMalloc((void **) &temp_gpu, (N * sizeof(float))));

    printf("image size: %d x %d\n", IMAGE_W, IMAGE_H);
    printf("%-#40s%-#20s%0s\n", "filter", "time", "FPS");
    printf("---------------------------------------------------------------------\n");


    CHECK_CUDA(hipDeviceSynchronize());
    auto timeStart = std::chrono::system_clock::now();
    gauss5x5_gpu_tiles(img_gpu, temp_gpu, IMAGE_W, IMAGE_H, cycles);
    CHECK_CUDA(hipDeviceSynchronize());  
    auto timeEnd = std::chrono::system_clock::now();
    double dProcessingTime = (double)std::chrono::duration_cast<std::chrono::milliseconds>(timeEnd - timeStart).count() / cycles;
    printTime("gauss5x5_gpu_tiles", dProcessingTime);



    CHECK_CUDA(hipDeviceSynchronize());
    timeStart = std::chrono::system_clock::now();
    gauss7x7_gpu_tiles(img_gpu, temp_gpu, IMAGE_W, IMAGE_H, cycles);
    CHECK_CUDA(hipDeviceSynchronize());  
    timeEnd = std::chrono::system_clock::now();
    dProcessingTime = (double)std::chrono::duration_cast<std::chrono::milliseconds>(timeEnd - timeStart).count() / cycles;
    printTime("gauss7x7_gpu_tiles", dProcessingTime);


    delete img_cpu;
    hipFree(img_gpu);
    hipFree(temp_gpu);

    return 0;
}