#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <sys/time.h>
#include <vector>
#define USECPSEC 1000000ULL
#define N 1024*1024

unsigned long long dtime_usec(unsigned long long start=0){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

__global__ void warmup_kernel(int *a, int *b, int *result, int n) {
    //Get the thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //Only calculate if thread index is valid
    if (tid < n) {
       int sum = 0;
       //Perform dot product calculation for this thread's segment of the arrays
       for (int i = tid; i < n; i+= blockDim.x * gridDim.x) {
           sum += a[i] * b[i];
       }
       //Atomically add this thread's result to the shared result
       atomicAdd(result, sum);
    }
}

__global__ void dotProductKernel2(int *a, int *b, int *results, int n) {
    //Get the thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //Only calculate if thread index is valid
    if (tid < n) {
       int sum = 0;
       //Perform dot product calculation for this thread's segment of the arrays
       for (int i = tid; i < n; i+= blockDim.x * gridDim.x) {
           sum += a[i] * b[i];
       }
       //Store this thread's result in the shared results array
       results[blockIdx.x * blockDim.x + threadIdx.x] = sum;
    } }

__global__ void sumResultsKernel(int *results, int *result, int n) {
    //Get the thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //Only calculate if thread index is valid
    if (tid == 0) {
       int sum = 0;
       //Sum up the partial results
       for (int i = 0; i < n; i++) {
           sum += results[i];
       }
       //Store the final result
       *result = sum;
    } 
}

void test_2() {
    int n = N;

    // Use vectors instead of raw pointers
    std::vector<int> a(n), b(n);
    int *c = (int*) malloc(sizeof(int));

    // Initialize the input vectors
    for (int i = 0; i < n; i++) {
        a[i] = 1; b[i] = 2;
    }
    *c = 0;

    // Determine the grid size and block size
    int blockSize = 1024;
    int gridSize = (n + blockSize - 1) / blockSize;

    // Allocate memory on the GPU
    int* d_a, *d_b, *d_results, *d_result;
    hipMalloc(&d_a, a.size() * sizeof(int));
    hipMalloc(&d_b, b.size() * sizeof(int));
    hipMalloc(&d_results, gridSize * blockSize * sizeof(int));
    hipMalloc(&d_result, 1 * sizeof(int));

    // Copy vectors to GPU
    hipMemcpy(d_a, a.data(), a.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), b.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_results, 0, gridSize * blockSize * sizeof(int));
    hipMemset(d_result, 0, sizeof(int));
    for(int i=0; i<10;i++){
      // Launch kernel
      dotProductKernel2<<<gridSize, blockSize>>>(d_a, d_b, d_results, n);
      // Sum up the partial results
      sumResultsKernel<<<1, 1024>>>(d_results, d_result, gridSize * blockSize);
      hipDeviceSynchronize();
    }

    // Copy result back from GPU
    hipMemcpy(c, d_result, 1 * sizeof(int),hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_results);
    hipFree(d_result);

    free(c); 
} 

int main(){
  //hipSetDevice(1);
  const int sz = N;
  int *a, *b, *c;
  hipMallocManaged(&a, sz*sizeof(a[0]));
  hipMallocManaged(&b, sz*sizeof(b[0]));
  hipMallocManaged(&c, sizeof(c[0]));
  for (int i = 0; i < sz; i++) {a[i] = 1; b[i] = 2;}
  c[0] = 0;
  hipMemPrefetchAsync(a, sz*sizeof(a[0]), 0);
  hipMemPrefetchAsync(b, sz*sizeof(b[0]), 0);
  hipMemPrefetchAsync(c, sizeof(c[0]), 0);
  // warm-up
  for(int i=0;i<3;i++){
    warmup_kernel<<<1, 1024>>>(a, b, c, sz);
    hipDeviceSynchronize();
  }
  test_2();

  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    std::cout << hipGetErrorString(err) << std::endl;
}