
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

const int my_bitset_size = 512;
const int my_bunch_size = 1024*2;
typedef unsigned uint;


__global__ void kernelXOR2(uint * bitset, uint * bunch, int * set_bits, int bitset_size, int bunch_size) {

    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if (tid < bunch_size){      // 1 Thread for each bitset in the 'bunch'
        int sum = 0;
        uint xor_res = 0;
        for (int i = 0; i < bitset_size; ++i){  // Iterate through every uint-block of the bitsets
            xor_res = bitset[i] ^ bunch[bitset_size * tid + i];
            sum += __popc(xor_res);
        }
        set_bits[tid] = sum;
    }
}

int test(){

// data setup

  uint *d_cbitset, *d_bitsets, *h_cbitset, *h_bitsets;
  int *d_r, *h_r, *h_ri;
  h_cbitset = new uint[my_bitset_size];
  h_bitsets = new uint[my_bitset_size*my_bunch_size];
  h_r  = new int[my_bunch_size];
  h_ri = new int[my_bunch_size];
  for (int i = 0; i < my_bitset_size*my_bunch_size; i++){
    h_bitsets[i] = rand();
    if (i < my_bitset_size) h_cbitset[i] = rand();}
  hipMalloc(&d_cbitset, my_bitset_size*sizeof(uint));
  hipMalloc(&d_bitsets, my_bitset_size*my_bunch_size*sizeof(uint));
  hipMalloc(&d_r,  my_bunch_size*sizeof(int));
  hipMemcpy(d_cbitset, h_cbitset, my_bitset_size*sizeof(uint), hipMemcpyHostToDevice);
  hipMemcpy(d_bitsets, h_bitsets, my_bitset_size*my_bunch_size*sizeof(uint), hipMemcpyHostToDevice);

  // Grid/Blocks used for kernel invocation
  dim3 block(32);
  dim3 grid((my_bunch_size-1)/32+1);

  kernelXOR2<<<grid, block>>>(d_cbitset, d_bitsets, d_r, my_bitset_size, my_bunch_size);
  hipMemcpy(h_r, d_r, my_bunch_size*sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < my_bunch_size; i++)
    if (h_r[i] != h_ri[i]) {std::cout << "mismatch at i: " << i << " was: " << h_ri[i] << " should be: " << h_r[i] << std::endl; return 0;}
  std::cout << "Results match." << std::endl;
  return 0;
}

int main(){
  hipSetDevice(1);
  for(int i = 0; i < 10; i++){
    test();
  }
  return 0;
}