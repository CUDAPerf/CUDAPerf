#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <sys/time.h>
#include <vector>
#define USECPSEC 1000000ULL
#define N 1024*1024

unsigned long long dtime_usec(unsigned long long start=0){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

__global__ void warmup_kernel(int *a, int *b, int *result, int n) {
    //Get the thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //Only calculate if thread index is valid
    if (tid < n) {
       int sum = 0;
       //Perform dot product calculation for this thread's segment of the arrays
       for (int i = tid; i < n; i+= blockDim.x * gridDim.x) {
           sum += a[i] * b[i];
       }
       //Atomically add this thread's result to the shared result
       atomicAdd(result, sum);
    }
}

__global__ void dotProductKernel1(int *a, int *b, int *result, int n)
{
    //Get the thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //Only calculate if thread index is valid
    if (tid < n) {
       int sum = 0;
       //Perform dot product calculation for this thread's segment of the arrays
       for (int i = tid; i < n; i+= blockDim.x * gridDim.x) {
           sum += a[i] * b[i];
       }
       //Atomically add this thread's result to the shared result
       atomicAdd(result, sum);
    }
}


int main(){
  //hipSetDevice(1);
  const int sz = N;
  int *a, *b, *c;
  hipMallocManaged(&a, sz*sizeof(a[0]));
  hipMallocManaged(&b, sz*sizeof(b[0]));
  hipMallocManaged(&c, sizeof(c[0]));
  for (int i = 0; i < sz; i++) {a[i] = 1; b[i] = 2;}
  c[0] = 0;
  hipMemPrefetchAsync(a, sz*sizeof(a[0]), 0);
  hipMemPrefetchAsync(b, sz*sizeof(b[0]), 0);
  hipMemPrefetchAsync(c, sizeof(c[0]), 0);
  // warm-up
  for(int i=0;i<3;i++){
    warmup_kernel<<<1, 1024>>>(a, b, c, sz);
    hipDeviceSynchronize();
  }
  for(int i=0;i<10;i++){
    dotProductKernel1<<<1, 1024>>>(a, b, c, sz);
    hipDeviceSynchronize();
  }

  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    std::cout << hipGetErrorString(err) << std::endl;
}