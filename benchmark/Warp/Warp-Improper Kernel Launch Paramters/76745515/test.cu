#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <sys/time.h>
#include <vector>
#define USECPSEC 1000000ULL
#define N 1024*1024

unsigned long long dtime_usec(unsigned long long start=0){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

__global__ void warmup_kernel(int *a, int *b, int *result, int n) {
    //Get the thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //Only calculate if thread index is valid
    if (tid < n) {
       int sum = 0;
       //Perform dot product calculation for this thread's segment of the arrays
       for (int i = tid; i < n; i+= blockDim.x * gridDim.x) {
           sum += a[i] * b[i];
       }
       //Atomically add this thread's result to the shared result
       atomicAdd(result, sum);
    }
}

__global__ void dotProductKernel1(int *a, int *b, int *result, int n)
{
    //Get the thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //Only calculate if thread index is valid
    if (tid < n) {
       int sum = 0;
       //Perform dot product calculation for this thread's segment of the arrays
       for (int i = tid; i < n; i+= blockDim.x * gridDim.x) {
           sum += a[i] * b[i];
       }
       //Atomically add this thread's result to the shared result
       atomicAdd(result, sum);
    }
}

__global__ void dotProductKernel2(int *a, int *b, int *results, int n) {
    //Get the thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //Only calculate if thread index is valid
    if (tid < n) {
       int sum = 0;
       //Perform dot product calculation for this thread's segment of the arrays
       for (int i = tid; i < n; i+= blockDim.x * gridDim.x) {
           sum += a[i] * b[i];
       }
       //Store this thread's result in the shared results array
       results[blockIdx.x * blockDim.x + threadIdx.x] = sum;
    } }

__global__ void sumResultsKernel(int *results, int *result, int n) {
    //Get the thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //Only calculate if thread index is valid
    if (tid == 0) {
       int sum = 0;
       //Sum up the partial results
       for (int i = 0; i < n; i++) {
           sum += results[i];
       }
       //Store the final result
       *result = sum;
    } 
}

const int BLOCK_SIZE=1024;
template <typename T>
__global__ void dotProductKernel(const T * __restrict__ a, const T * __restrict__ b, T * __restrict__ result, size_t n)
{
    __shared__ T smem[BLOCK_SIZE];
    //Get the thread index
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    T sum = 0;
    //Perform dot product calculation for this thread's segment of the arrays
    for (size_t i = tid; i < n; i+= gridDim.x*blockDim.x) {
      sum += a[i] * b[i];
      }
    //Perform threadblock sweep reduction
    smem[threadIdx.x] = sum;
    for (int i = BLOCK_SIZE>>1; i > 0; i>>=1){
      __syncthreads();
      if (threadIdx.x < i) smem[threadIdx.x] += smem[threadIdx.x+i];}
    //Atomically add this block's result to the global result
    if ((!threadIdx.x) && (tid < n)) atomicAdd(result, smem[0]);
}


void test_2() {
    int n = N;

    // Use vectors instead of raw pointers
    std::vector<int> a(n), b(n);
    int *c = (int*) malloc(sizeof(int));

    // Initialize the input vectors
    for (int i = 0; i < n; i++) {
        a[i] = 1; b[i] = 2;
    }
    *c = 0;

    // Determine the grid size and block size
    int blockSize = 1024;
    int gridSize = (n + blockSize - 1) / blockSize;

    // Allocate memory on the GPU
    int* d_a, *d_b, *d_results, *d_result;
    hipMalloc(&d_a, a.size() * sizeof(int));
    hipMalloc(&d_b, b.size() * sizeof(int));
    hipMalloc(&d_results, gridSize * blockSize * sizeof(int));
    hipMalloc(&d_result, 1 * sizeof(int));

    // Copy vectors to GPU
    hipMemcpy(d_a, a.data(), a.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), b.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_results, 0, gridSize * blockSize * sizeof(int));
    hipMemset(d_result, 0, sizeof(int));
    for(int i=0; i<10;i++){
      // Launch kernel
      dotProductKernel2<<<gridSize, blockSize>>>(d_a, d_b, d_results, n);
      // Sum up the partial results
      sumResultsKernel<<<1, 1024>>>(d_results, d_result, gridSize * blockSize);
      hipDeviceSynchronize();
    }

    // Copy result back from GPU
    hipMemcpy(c, d_result, 1 * sizeof(int),hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_results);
    hipFree(d_result);

    free(c); 
} 

int main(){
  //hipSetDevice(1);
  const int sz = N;
  int *a, *b, *c;
  hipMallocManaged(&a, sz*sizeof(a[0]));
  hipMallocManaged(&b, sz*sizeof(b[0]));
  hipMallocManaged(&c, sizeof(c[0]));
  for (int i = 0; i < sz; i++) {a[i] = 1; b[i] = 2;}
  c[0] = 0;
  hipMemPrefetchAsync(a, sz*sizeof(a[0]), 0);
  hipMemPrefetchAsync(b, sz*sizeof(b[0]), 0);
  hipMemPrefetchAsync(c, sizeof(c[0]), 0);
  // warm-up
  for(int i=0;i<3;i++){
    warmup_kernel<<<1, 1024>>>(a, b, c, sz);
    hipDeviceSynchronize();
  }
  for(int i=0;i<10;i++){
    dotProductKernel1<<<1, 1024>>>(a, b, c, sz);
    hipDeviceSynchronize();
  }

  for(int i=0;i<10;i++){
    dotProductKernel<<<128, BLOCK_SIZE>>>(a, b, c, sz);
    hipDeviceSynchronize();
  }
  test_2();

  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    std::cout << hipGetErrorString(err) << std::endl;
}