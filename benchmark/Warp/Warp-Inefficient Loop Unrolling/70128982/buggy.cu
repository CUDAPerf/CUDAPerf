#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/copy.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>

template <typename T>
void cpufunction(T *result, T *oldArray, size_t size, T k) {
    for (int i = 2; i < size; i++) {
        result[i] = oldArray[i] + k * result[i - 2];
    }
}

struct scan_op // as per blelloch (1.7)
{
    template <typename T1, typename T2>
    __host__ __device__
    T1 operator()(const T1 &t1, const T2 &t2) {
        T1 ret;
        thrust::get<0>(ret) = thrust::get<0>(t1) * thrust::get<2>(t2) + thrust::get<1>(t1) * thrust::get<4>(t2) + thrust::get<0>(t2);
        thrust::get<1>(ret) = thrust::get<0>(t1) * thrust::get<3>(t2) + thrust::get<1>(t1) * thrust::get<5>(t2) + thrust::get<1>(t2);
        thrust::get<2>(ret) = thrust::get<2>(t1) * thrust::get<2>(t2) + thrust::get<3>(t1) * thrust::get<4>(t2);
        thrust::get<3>(ret) = thrust::get<2>(t1) * thrust::get<3>(t2) + thrust::get<3>(t1) * thrust::get<5>(t2);
        thrust::get<4>(ret) = thrust::get<4>(t1) * thrust::get<2>(t2) + thrust::get<5>(t1) * thrust::get<4>(t2);
        thrust::get<5>(ret) = thrust::get<4>(t1) * thrust::get<3>(t2) + thrust::get<5>(t1) * thrust::get<5>(t2);
        return ret;
    }
};

typedef float mt;
const size_t ds = 1024*1024/8;
const mt k = 1.01;
const int snip = 10;

int main() {
    mt *b1  = new mt[ds]; // b as in blelloch (1.5)
    mt *cr = new mt[ds]; // cpu result
    for (int i = 0; i < ds; i++) { b1[i] = rand() / (float)RAND_MAX; }
    cr[0] = b1[0];
    cr[1] = b1[1];

    // Measure CPU time
    hipEvent_t start_cpu, stop_cpu;
    hipEventCreate(&start_cpu);
    hipEventCreate(&stop_cpu);
    hipEventRecord(start_cpu, 0);

    cpufunction(cr, b1, ds, k);

    hipEventRecord(stop_cpu, 0);
    hipEventSynchronize(stop_cpu);
    float cpu_time = 0;
    hipEventElapsedTime(&cpu_time, start_cpu, stop_cpu);

    for (int i = 0; i < snip; i++) std::cout << cr[i] << ",";
    for (int i = ds - snip; i < ds; i++) std::cout << cr[i] << ",";
    std::cout << std::endl;

    // Print the times
    std::cout << "CPU Time: " << cpu_time << " ms" << std::endl;

    // Cleanup
    delete[] b1;
    delete[] cr;
    hipEventDestroy(start_cpu);
    hipEventDestroy(stop_cpu);

    return 0;
}